
#include <hip/hip_runtime.h>
__global__ void update_src( int Nx, int Ny, int Nz, int tstep, float *F ) {
	int idx, ijk;
	idx = threadIdx.x;
	//ijk = (idx+1)*Ny*Nz + (Ny/2)*Nz + (Nz/2);
	//ijk = (idx+1)*Ny*Nz + (Ny/2 - 30)*Nz + (Nz/2 - 50);
	//ijk = (Nx/2 - 30)*Ny*Nz + (idx)*Nz + (Nz/2 - 50);
	ijk = (Nx/2-30)*Ny*Nz + (Ny/2-50)*Nz + idx;

	F[ijk] += sin(0.1*tstep);
}

