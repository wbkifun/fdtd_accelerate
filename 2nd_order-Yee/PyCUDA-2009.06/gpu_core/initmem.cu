
#include <hip/hip_runtime.h>
__global__ void initmem( int Ntot, float *a ) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	if ( idx < Ntot ) a[idx] = 0;
}
