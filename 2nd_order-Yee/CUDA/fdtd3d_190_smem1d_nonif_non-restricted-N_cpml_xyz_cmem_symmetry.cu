#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdarg.h>
#include <math.h>
#include <hdf5.h>

#define NPML 10

const float light_velocity = 2.99792458e8;	// m s- 
const float ep0 = 8.85418781762038920e-12;	// F m-1 (permittivity at vacuum)
const float	mu0 = 1.25663706143591730e-6;	// N A-2 (permeability at vacuum)
const float imp0 = sqrt( mu0/ep0 );	// (impedance at vacuum)
const float pi = 3.14159265358979323846;

const int MBPG = 65535;

// Allocate constant memory for CPML
__constant__ float rcmbE[2*NPML+1];
__constant__ float rcmaE[2*NPML+1];
__constant__ float rcmbH[2*NPML+1];
__constant__ float rcmaH[2*NPML+1];


typedef struct N3 {
	int x, y, z;
} N3;


typedef struct P3F3 {
	float ***x, ***y, ***z;
} P3F3;


typedef struct P1F3 {
	float *x, *y, *z;
} P1F3;


typedef struct P1F2 {
	float *f, *b;
} P1F2;


typedef struct P1F6 {
	P1F2 x, y, z;
} P1F6;


__host__ void updateTimer(time_t t0, int tstep, char str[]) {
	int elapsedTime=(int)(time(0)-t0);
	sprintf(str, "%02d:%02d:%02d", elapsedTime/3600, elapsedTime%3600/60, elapsedTime%60);
}


__host__ void exec(char *format, ...) {
	char str[1024];
	va_list ap;
	va_start(ap, format);
	vsprintf(str, format, ap);
	system(str);
}


__host__ void dumpToH5(int Ni, int Nj, int Nk, int is, int js, int ks, int ie, int je, int ke, float ***f, char *format, ...) {
	char filename[1024];
	va_list ap;
	va_start(ap, format);
	vsprintf(filename, format, ap);
	hid_t file, dataset, filespace, memspace;

	hsize_t dimsm[3] = { Ni, Nj, Nk };
	hsize_t start[3] = { is, js, ks };
	hsize_t count[3] = { 1-is+ie, 1-js+je, 1-ks+ke };
	memspace = H5Screate_simple(3, dimsm, 0);
	filespace = H5Screate_simple(3, count, 0);
	file = H5Fcreate(filename, H5F_ACC_TRUNC, H5P_DEFAULT, H5P_DEFAULT);
	dataset = H5Dcreate(file, "Data", H5T_NATIVE_FLOAT, filespace, H5P_DEFAULT);
	H5Sselect_hyperslab(memspace, H5S_SELECT_SET, start, 0, count, 0);
	H5Dwrite(dataset, H5T_NATIVE_FLOAT, memspace, filespace, H5P_DEFAULT, f[0][0]);
	H5Dclose(dataset);
	H5Sclose(filespace);
	H5Sclose(memspace);
	H5Fclose(file);
}


__host__ float ***makeArray3D(int Nx, int Ny, int Nz) {
	float ***f;
	int i;

	f = (float ***) calloc (Nx, sizeof(float **));
	f[0] = (float **) calloc (Ny*Nx, sizeof(float *));
	f[0][0] = (float *) calloc (Nz*Ny*Nx, sizeof(float));

	for (i=0; i<Nx; i++) f[i] = f[0] + i*Ny;
	for (i=0; i<Ny*Nx; i++) f[0][i] = f[0][0] + i*Nz;

	return f;
}


__host__ float **makeArray2D(int Nx, int Ny) {
	float **f;

	f = (float **) calloc (Nx, sizeof(float *));
	f[0] = (float *) calloc (Ny*Nx, sizeof(float));

	for (int i=0; i<Nx; i++) f[i] = f[0] + i*Ny;

	return f;
}


__host__ float *makeArray1D( int Nx ) {
	float *f;

	f = (float *) calloc (Nx, sizeof(float));

	return f;
}


__host__ void set_geometry( N3 N, P3F3 CE ) {
	int i,j,k;

	for ( i=1; i<N.x; i++ ) {
		for ( j=1; j<N.y; j++ ) {
			for ( k=1; k<N.z; k++ ) {
				CE.x[i][j][k] = 0.5;
				CE.y[i][j][k] = 0.5;
				CE.z[i][j][k] = 0.5;

				if ( i == N.x-1 ) {
					CE.y[i][j][k] = 0;
					CE.z[i][j][k] = 0;
				}
				if ( j == N.y-1 ) {
					CE.z[i][j][k] = 0;
					CE.x[i][j][k] = 0;
				}
				if ( k == N.z-1 ) {
					CE.x[i][j][k] = 0;
					CE.y[i][j][k] = 0;
				}
			}
		}
	}
	/*
	int idx;
	for ( idx=0; idx<(N.x+1)*N.y*N.z; idx++ ) {
		i = idx/(N.y*N.z);
		j = ( idx - i*N.y*N.z )/N.z;
		k = idx%N.z;
		printf("%d [%d,%d,%d] %g, %g, %g\n", idx, i, j, k, CE.x[0][0][idx], CE.y[0][0][idx], CE.z[0][0][idx] );
	}
	*/
}


__host__ void verify_16xNz(int Nz) {
	int R = Nz%16;
	int N1 = Nz-R; 
	int N2 = N1+16; 
	if ( R == 0 ) printf("Nz is a multiple of 16.\n");
	else {
		printf("Error: Nz is not a multiple of 16.\n");
		printf("Recommend Nz: %d or %d\n", N1, N2);
		exit(0);
	}
}


__host__ float calcOccupancy(int TPB) {
	float occupancy;
	int WPB;	// wrap/block
	int ABPM;	// active block/streaming multiprocessor
	int AWPM;	// active warp/streaming multiprocessor
	int MAX_ABPM = 8;	 
	int MAX_AWPM = 32;	 
	//int MAX_TPM = 1024;	 
	int TPW = 32;	// thread/warp

	WPB = TPB%TPW == 0 ? TPB/TPW : TPB/TPW+1;
	ABPM = MAX_AWPM/WPB < MAX_ABPM ? MAX_AWPM/WPB : MAX_ABPM;
	AWPM = WPB*ABPM;
	occupancy = (float)AWPM/MAX_AWPM;

	return occupancy;
}


__host__ int selectTPB(int Ntot, int Nsurplus_plane) {
	int i;
	int *tpb, bpg, TPB=0;
	int Nsurplus;
	float occupancy, max_occupancy=0;

	int Ntpb = 512/16 + 2;
	tpb = (int *) calloc (Ntpb, sizeof(int));
	tpb[0] = 512;
	tpb[1] = 256;
	tpb[2] = 128;
	for ( i=3; i<Ntpb; i++ ) tpb[i] = tpb[0] - 16*(i-2);
	//for ( i=0; i<Ntpb; i++ ) printf("tpb[%d]=%d\n",i,tpb[i]);
	
	for ( i=0; i<Ntpb; i++) {
		occupancy = calcOccupancy( tpb[i] );
		if ( occupancy > max_occupancy ) {
			max_occupancy = occupancy;
			bpg = Ntot%tpb[i] == 0 ? Ntot/tpb[i] : Ntot/tpb[i] + 1;
			Nsurplus = tpb[i]*bpg - Ntot;
			if ( Nsurplus_plane == 0 )  TPB = tpb[i];
			else if ( Nsurplus <= Nsurplus_plane ) TPB = tpb[i];
		}
	}

	if ( TPB == 0 ) {
		printf("Error: There is not a TPB satisfied the conditions\n");
		exit(0);
	}

	printf("\tNsurplus_plane=%d, Nsurplus=%d\n", Nsurplus_plane, Nsurplus);
	printf("\tNtot=%d, TPB=%d\n", Ntot, TPB);

	return TPB;
}


__global__ void initArray(int Ntot, float *a, int idx0) {
	int idx = idx0 + blockIdx.x*blockDim.x + threadIdx.x;

	if ( idx < Ntot ) a[idx] = 0;
}


__host__ void initMainArrays(int Ntot, P1F3 F) {
	int i;
	int TPB, BPG, NK, sBPG, *idx0;
	dim3 Db, *Dg;

	printf("select TPB,BPG: main init\n");
	TPB = 512;
	Db = dim3( TPB );
	BPG = Ntot%TPB == 0 ? Ntot/TPB : Ntot/TPB + 1;
	NK = BPG/MBPG + 1;	// Number of kernel
	sBPG = BPG/NK;
	Dg = (dim3 *) malloc ( NK*sizeof(dim3) );
	idx0 = (int *) malloc ( NK*sizeof(int) );
	for ( i=0; i<NK; i++ ) {
		idx0[i] = TPB*sBPG*i;
		Dg[i] = dim3(sBPG);
	}
	Dg[NK-1] = dim3(sBPG+BPG%NK);
	printf("\tTPB=%d, BPG=%d, sBPG(%d)=%d\n", TPB, BPG, NK, sBPG);

	for ( i=0; i<NK; i++ ) {
		initArray <<<Dg[i],Db>>> (Ntot, F.x, idx0[i]); 
		initArray <<<Dg[i],Db>>> (Ntot, F.y, idx0[i]); 
		initArray <<<Dg[i],Db>>> (Ntot, F.z, idx0[i]); 
	}
}


__host__ void initPsiArrays(int Ntot, dim3 Db, dim3 Dg, P1F2 psi1, P1F2 psi2) {
	initArray <<<Dg,Db>>> (Ntot, psi1.f, 0); 
	initArray <<<Dg,Db>>> (Ntot, psi1.b, 0); 
	initArray <<<Dg,Db>>> (Ntot, psi2.f, 0); 
	initArray <<<Dg,Db>>> (Ntot, psi2.b, 0); 
}


__host__ void freeMainArrays(P1F3 F) {
	hipFree(F.x);
	hipFree(F.y);
	hipFree(F.z);
}


__host__ void freePsiArrays(P1F6 psix, P1F6 psiy, P1F6 psiz) {
	hipFree(psix.y.f);
	hipFree(psix.y.b);
	hipFree(psix.z.f);
	hipFree(psix.z.b);

	hipFree(psiy.z.f);
	hipFree(psiy.z.b);
	hipFree(psiy.x.f);
	hipFree(psiy.x.b);

	hipFree(psiz.x.f);
	hipFree(psiz.x.b);
	hipFree(psiz.y.f);
	hipFree(psiz.y.b);
}


__global__ void updateE(N3 N, P1F3 E, P1F3 H, P1F3 CE, int idx0) {
	int tk = threadIdx.x;
	int idx = blockIdx.x*blockDim.x + tk + idx0;
	int Nyz = N.y*N.z;
	int eidx = idx + Nyz;

	extern __shared__ float hs[];
	float* hx = (float*) hs;
	float* hy = (float*) &hx[blockDim.x+1];
	float* hz = (float*) &hy[blockDim.x+1];

	hx[tk] = H.x[idx];
	hy[tk] = H.y[idx];
	hz[tk] = H.z[idx];
	
	if ( tk==blockDim.x-1 ) {
		hx[tk+1] = H.x[idx+1];
		hy[tk+1] = H.y[idx+1];
	}
	__syncthreads();

	E.x[eidx] += CE.x[idx]*( H.z[idx+N.z] - hz[tk] - hy[tk+1] + hy[tk] );
	E.y[eidx] += CE.y[idx]*( hx[tk+1] - hx[tk] - H.z[idx+Nyz] + hz[tk] );
	E.z[eidx] += CE.z[idx]*( H.y[idx+Nyz] - hy[tk] - H.x[idx+N.z] + hx[tk] );
}


__global__ void updateH(N3 N, P1F3 E, P1F3 H, int idx0) {
	int tk = threadIdx.x;
	int idx = blockIdx.x*blockDim.x + tk + idx0;
	int Nyz = N.y*N.z;
	int eidx = idx + Nyz;

	extern __shared__ float es[];
	float* ex = (float*) es;
	float* ey = (float*) &ex[blockDim.x+1];
	float* ez = (float*) &ey[blockDim.x+1];

	ex[tk+1] = E.x[eidx];
	ey[tk+1] = E.y[eidx];
	ez[tk] = E.z[eidx];
	if ( tk==0 ) {
		ex[0] = E.x[eidx-1];
		ey[0] = E.y[eidx-1];
	}
	__syncthreads();

	H.x[idx] -= 0.5*( ez[tk] - E.z[eidx-N.z] - ey[tk+1] + ey[tk] );
	H.y[idx] -= 0.5*( ex[tk+1] - ex[tk] - ez[tk] + E.z[eidx-Nyz] );
	H.z[idx] -= 0.5*( ey[tk+1] - E.y[eidx-Nyz] - ex[tk+1] + E.x[eidx-N.z] );
}


__global__ void updateSrc(N3 N, P1F3 E, int tstep) {
	int idx = threadIdx.x;
	int ijk = (idx+1)*N.y*N.z + (N.y/2)*N.z + (N.z/2);
	//int ijk = (idx+1)*N.y*N.z + (N.y/2)*N.z + (N.z/2 - 50);
	//int ijk = (N.x/2+1)*N.y*N.z + (N.y/2)*N.z + idx;

	E.x[ijk] += sin(0.1*tstep);
	//E.z[ijk] += sin(0.1*tstep);
}


__global__ void updateCPMLxE(N3 N, P1F3 E, P1F3 H, P1F3 CE, float *psi1, float *psi2, int backward) {
	int pidx = blockIdx.x*blockDim.x + threadIdx.x;
	int Nyz = N.y*N.z;
	int pi = pidx/Nyz + backward*(NPML+1);

	int idx = pidx + ( 1 + backward*(N.x-NPML-2) )*Nyz;
	int eidx = idx + Nyz;

	psi1[pidx] = rcmbE[pi]*psi1[pidx] + rcmaE[pi]*( H.z[idx+Nyz] - H.z[idx] );
	E.y[eidx] -= CE.y[idx]*psi1[pidx];

	psi2[pidx] = rcmbE[pi]*psi2[pidx] + rcmaE[pi]*( H.y[idx+Nyz] - H.y[idx] );
	E.z[eidx] += CE.z[idx]*psi2[pidx];
}


__global__ void updateCPMLxH(N3 N, P1F3 E, P1F3 H, float *psi1, float *psi2, int backward) {
	int pidx = blockIdx.x*blockDim.x + threadIdx.x;
	int Nyz = N.y*N.z;
	int pi = pidx/Nyz + backward*(NPML+1);

	int idx = pidx + ( 1 + backward*(N.x-NPML-1) )*Nyz;
	int eidx = idx + Nyz;

	psi1[pidx] = rcmbH[pi]*psi1[pidx] + rcmaH[pi]*( E.z[eidx] - E.z[eidx-Nyz] );
	H.y[idx] += 0.5*psi1[pidx];

	psi2[pidx] = rcmbH[pi]*psi2[pidx] + rcmaH[pi]*( E.y[eidx] - E.y[eidx-Nyz] );
	H.z[idx] -= 0.5*psi2[pidx];
}


__global__ void updateCPMLyE(N3 N, P1F3 E, P1F3 H, P1F3 CE, float *psi1, float *psi2, int backward) {
	int pidx = blockIdx.x*blockDim.x + threadIdx.x;
	int i = pidx/(NPML*N.z);
	int pj = ( pidx/N.z )%NPML + backward*(NPML+1);

	int idx = pidx + ( 1 + i*(N.y-NPML) + backward*(N.y-NPML-2) )*N.z;
	int eidx = idx + N.y*N.z;

	psi1[pidx] = rcmbE[pj]*psi1[pidx] + rcmaE[pj]*( H.x[idx+N.z] - H.x[idx] );
	E.z[eidx] -= CE.z[idx]*psi1[pidx];

	psi2[pidx] = rcmbE[pj]*psi2[pidx] + rcmaE[pj]*( H.z[idx+N.z] - H.z[idx] );
	E.x[eidx] += CE.x[idx]*psi2[pidx];
}


__global__ void updateCPMLyH(N3 N, P1F3 E, P1F3 H, float *psi1, float *psi2, int backward) {
	int pidx = blockIdx.x*blockDim.x + threadIdx.x;
	int i = pidx/(NPML*N.z);
	int pj = ( pidx/N.z )%NPML + backward*(NPML+1);

	int idx = pidx + ( 1 + i*(N.y-NPML) + backward*(N.y-NPML-1) )*N.z;
	int eidx = idx + N.y*N.z;

	psi1[pidx] = rcmbH[pj]*psi1[pidx] + rcmaH[pj]*( E.x[eidx] - E.x[eidx-N.z] );
	H.z[idx] += 0.5*psi1[pidx];

	psi2[pidx] = rcmbH[pj]*psi2[pidx] + rcmaH[pj]*( E.z[eidx] - E.z[eidx-N.z] );
	H.x[idx] -= 0.5*psi2[pidx];
}


__global__ void updateCPMLzE(N3 N, P1F3 E, P1F3 H, P1F3 CE, float *psi1, float *psi2, int backward) {
	int tk = threadIdx.x;
	int pidx = blockIdx.x*blockDim.x + tk;
	int j = pidx/NPML;
	int pk = pidx%NPML + backward*(NPML+1);

	int idx = pidx + 1 + j*(N.z-NPML) + backward*(N.z-NPML-2);
	int eidx = idx + N.y*N.z;

	extern __shared__ float hs[];
	float* hx = (float*) hs;
	float* hy = (float*) &hx[blockDim.x+1];

	hx[tk] = H.x[idx];
	hy[tk] = H.y[idx];
	
	if ( tk == blockDim.x-1 ) {
		hx[tk+1] = H.x[idx+1];
		hy[tk+1] = H.y[idx+1];
	}
	__syncthreads();

	//psi1[pidx] = rcmbE[pk]*psi1[pidx] + rcmaE[pk]*( hy[tk+1] - hy[tk] );
	psi1[pidx] = rcmbE[pk]*psi1[pidx] + rcmaE[pk]*( H.y[idx+1] - H.y[idx] );
	E.x[eidx] -= CE.x[idx]*psi1[pidx];

	//psi2[pidx] = rcmbE[pk]*psi2[pidx] + rcmaE[pk]*( hx[tk+1] - hx[tk] );
	psi2[pidx] = rcmbE[pk]*psi2[pidx] + rcmaE[pk]*( H.x[idx+1] - H.x[idx] );
	E.y[eidx] += CE.y[idx]*psi2[pidx];
	/*
	//printf("tk=%d, pidx=%d, pk=%d, idx=%d, eidx=%d, CEx=%g, Ex=%g, CEy=%g, Ey=%g\n", tk, pidx, pk, idx, eidx, CE.x[idx], E.x[eidx], CE.y[idx], E.y[eidx]);
	int i = idx/(N.y*N.z);
	int j = (idx - i*N.y*N.z)/N.z;
	int k = idx%N.z;
	//printf("[%d,%d,%d] [%d,%d,%d,%d,%d] [%g, %g, %g, %g]\n", i,j,k,tk, pidx, pk, idx, eidx, CE.x[idx], E.x[eidx], CE.y[idx], E.y[eidx]);
	printf("[%d,%d,%d] [%d,%d,%d,%d,%d] [%g, %g]\n", i,j,k,tk, pidx, pk, idx, eidx, CE.x[idx], CE.y[idx]);
	*/
}


__global__ void updateCPMLzH(N3 N, P1F3 E, P1F3 H, float *psi1, float *psi2, int backward) {
	int tk = threadIdx.x;
	int pidx = blockIdx.x*blockDim.x + tk;
	int j = pidx/NPML;
	int pk = pidx%NPML + backward*(NPML+1);

	int idx = pidx + 1 + j*(N.z-NPML) + backward*(N.z-NPML-1);
	int eidx = idx + N.y*N.z;

	extern __shared__ float es[];
	float* ex = (float*) es;
	float* ey = (float*) &ex[blockDim.x+1];

	ex[tk+1] = E.x[eidx];
	ey[tk+1] = E.y[eidx];
	if ( tk == 0 ) {
		ex[0] = E.x[eidx-1];
		ey[0] = E.y[eidx-1];
	}
	__syncthreads();

	//psi1[pidx] = rcmbH[pk]*psi1[pidx] + rcmaH[pk]*( ey[tk+1] - ey[tk] );
	psi1[pidx] = rcmbH[pk]*psi1[pidx] + rcmaH[pk]*( E.y[eidx] - E.y[eidx-1] );
	H.x[idx] += 0.5*psi1[pidx];

	//psi2[pidx] = rcmbH[pk]*psi2[pidx] + rcmaH[pk]*( ex[tk+1] - ex[tk] );
	psi2[pidx] = rcmbH[pk]*psi2[pidx] + rcmaH[pk]*( E.x[eidx] - E.x[eidx-1] );
	H.y[idx] -= 0.5*psi2[pidx];
}



int main() {
	int tstep;
	char time_str[32];
	time_t t0;
	int i;

	// --------------------------------------------------------------------------------
	// Set the parameters
	N3 N;
	N.x = 200;
	N.y = 200;
	N.z = 304;
	int TMAX = 500;
	//N.x = 20;
	//N.y = 20;
	//N.z = 32;
	//int TMAX = 1;
	
	float S = 0.5;
	float dx = 10e-9;
	float dt = S*dx/light_velocity;

	int Npml = NPML;
	printf("N(%d,%d,%d), TMAX=%d\n", N.x, N.y, N.z, TMAX);
	verify_16xNz( N.z );
	printf("Npml=%d\n",Npml);

	// --------------------------------------------------------------------------------
	// Allocate host memory
	P3F3 CE;
	CE.x = makeArray3D( N.x+1, N.y, N.z );
	CE.y = makeArray3D( N.x+1, N.y, N.z );
	CE.z = makeArray3D( N.x+1, N.y, N.z );

	//float **Ex, **Ez;
	//Ex = makeArray2D( N.y, N.z );
	//Ez = makeArray2D( N.x, N.y );
	float ***Ex, ***Ey, ***Ez;
	Ex = makeArray3D( N.x+2, N.y, N.z );
	Ey = makeArray3D( N.x+2, N.y, N.z );
	Ez = makeArray3D( N.x+2, N.y, N.z );
	float ***Hx, ***Hy, ***Hz;
	Hx = makeArray3D( N.x+2, N.y, N.z );
	Hy = makeArray3D( N.x+2, N.y, N.z );
	Hz = makeArray3D( N.x+2, N.y, N.z );

	// --------------------------------------------------------------------------------
	// Geometry
	set_geometry( N, CE );

	// --------------------------------------------------------------------------------
	// Parameters for CPML
	int m = 4;	// grade_order
	float sigma_max = (m+1.)/(15*pi*Npml*dx);
	float alpha = 0.05;
	float *sigmaE, *bE, *aE;
	float *sigmaH, *bH, *aH;

	sigmaE = (float *) calloc (2*Npml+1, sizeof(float));
	sigmaH = (float *) calloc (2*Npml+1, sizeof(float));
	bE = (float *) calloc (2*Npml+1, sizeof(float));
	bH = (float *) calloc (2*Npml+1, sizeof(float));
	aE = (float *) calloc (2*Npml+1, sizeof(float));
	aH = (float *) calloc (2*Npml+1, sizeof(float));
	for (i=0; i<Npml; i++) {
		sigmaE[i] = pow( (Npml-0.5-i)/Npml, m )*sigma_max;
		sigmaE[i+Npml+1] = pow( (0.5+i)/Npml, m )*sigma_max;
		sigmaH[i] = pow( (float)(Npml-i)/Npml, m )*sigma_max;
		sigmaH[i+Npml+1] = pow( (1.+i)/Npml, m )*sigma_max;
	}

	for (i=0; i<2*Npml+1; i++) {
		bE[i] = exp( -(sigmaE[i] + alpha)*dt/ep0 );
		bH[i] = exp( -(sigmaH[i] + alpha)*dt/ep0 );
		aE[i] = sigmaE[i]/(sigmaE[i]+alpha)*(bE[i]-1);
		aH[i] = sigmaH[i]/(sigmaH[i]+alpha)*(bH[i]-1);
		//printf("[%d]\tsigmaE=%g,\tbE=%g,aE=%g\n", i, sigmaE[i], bE[i], aE[i]);
		//printf("[%d]\tsigmaH=%g,\tbH=%g,aH=%g\n", i, sigmaH[i], bH[i], aH[i]);
	}

	free(sigmaE);
	free(sigmaH);

	// --------------------------------------------------------------------------------
	// Copy arrays from host to constant memory
	hipMemcpyToSymbol(HIP_SYMBOL(rcmbE), bE, (2*Npml+1)*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(rcmaE), aE, (2*Npml+1)*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(rcmbH), bH, (2*Npml+1)*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(rcmaH), aH, (2*Npml+1)*sizeof(float));

	free(bE);
	free(aE);
	free(bH);
	free(aH);

	// --------------------------------------------------------------------------------
	// Set the GPU parameters
	// TPB: Number of threads per block
	// BPG: Number of thread blocks per grid
	int Ntot, TPB, BPG;
	int NK, sBPG, *idx0;

	// main update
	printf("select TPB,BPG: main\n");
	dim3 Db_main, *Dg_main;
	Ntot = N.x*N.y*N.z;
	TPB = selectTPB( Ntot, N.y*N.z );
	Db_main = dim3( TPB );
	BPG = Ntot%TPB == 0 ? Ntot/TPB : Ntot/TPB + 1;
	NK = BPG/MBPG + 1;	// Number of kernel
	sBPG = BPG/NK;
	Dg_main = (dim3 *) malloc ( NK*sizeof(dim3) );
	idx0 = (int *) malloc ( NK*sizeof(int) );
	for ( i=0; i<NK; i++ ) {
		idx0[i] = TPB*sBPG*i;
		Dg_main[i] = dim3(sBPG);
	}
	Dg_main[NK-1] = dim3(sBPG+BPG%NK);
	size_t Ns_main = sizeof(float)*( 2*(TPB+1)+TPB );
	printf("\tBPG=%d, sBPG(%d)=%d, Ns_main=%d\n", BPG, NK, sBPG, Ns_main);
	
	// source 
	TPB = N.x;
	//TPB = N.z;
	BPG = 1;
	dim3 DBsrc(TPB);
	dim3 DGsrc(BPG);
	printf("source: TPB=%d, BPG=%d\n", TPB, BPG);

	// cpml 
	printf("select TPB,BPG: pml x\n");
	dim3 Db_pmlx, Dg_pmlx;
	Ntot = Npml*N.y*N.z;
	TPB = selectTPB( Ntot, N.y*N.z );
	BPG = Ntot%TPB == 0 ? Ntot/TPB : Ntot/TPB + 1;
	Db_pmlx = dim3( TPB );
	Dg_pmlx = dim3( BPG );
	int Ntotpmlx = TPB*BPG;
	printf("\tBPG=%d\n", BPG);

	printf("select TPB,BPG: pml y\n");
	dim3 Db_pmly, Dg_pmly;
	Ntot = N.x*Npml*N.z;
	TPB = selectTPB( Ntot, Npml*N.z );
	BPG = Ntot%TPB == 0 ? Ntot/TPB : Ntot/TPB + 1;
	Db_pmly = dim3( TPB );
	Dg_pmly = dim3( BPG );
	int Ntotpmly = TPB*BPG;
	printf("\tBPG=%d\n", BPG);

	printf("select TPB,BPG: pml z\n");
	dim3 Db_pmlz, Dg_pmlz;
	//int Npml_pitch = (Npml/16 + 1)*16; 
	Ntot = N.x*N.y*Npml;
	TPB = selectTPB( Ntot, N.y*Npml );
	BPG = Ntot%TPB == 0 ? Ntot/TPB : Ntot/TPB + 1;
	Db_pmlz = dim3( TPB );
	Dg_pmlz = dim3( BPG );
	int Ntotpmlz = TPB*BPG;
	size_t Ns_pmlz = sizeof(float)*( 2*(TPB+1) );
	printf("\tBPG=%d, Ns_pmlz=%d\n", BPG, Ns_pmlz);


	// --------------------------------------------------------------------------------
	// Allocate device memory
	P1F3 devE, devH;
	P1F3 devCE;

	int size_devF = (N.x+2)*N.y*N.z*sizeof(float);
	int size_devC = (N.x+1)*N.y*N.z*sizeof(float);

	hipMalloc ( (void**) &devE.x, size_devF );
	hipMalloc ( (void**) &devE.y, size_devF );
	hipMalloc ( (void**) &devE.z, size_devF );
	hipMalloc ( (void**) &devH.x, size_devF );
	hipMalloc ( (void**) &devH.y, size_devF );
	hipMalloc ( (void**) &devH.z, size_devF );
	hipMalloc ( (void**) &devCE.x, size_devC );
	hipMalloc ( (void**) &devCE.y, size_devC );
	hipMalloc ( (void**) &devCE.z, size_devC );
	
	// --------------------------------------------------------------------------------
	// Allocate device memory for CPML
	P1F6 psixE, psiyE, psizE;
	P1F6 psixH, psiyH, psizH;

	int size_psix = Ntotpmlx*sizeof(float);
	int size_psiy = Ntotpmly*sizeof(float);
	int size_psiz = Ntotpmlz*sizeof(float);

	hipMalloc ( (void**) &psixE.y.f, size_psix );
	hipMalloc ( (void**) &psixE.y.b, size_psix );
	hipMalloc ( (void**) &psixE.z.f, size_psix );
	hipMalloc ( (void**) &psixE.z.b, size_psix );
	hipMalloc ( (void**) &psixH.y.f, size_psix );
	hipMalloc ( (void**) &psixH.y.b, size_psix );
	hipMalloc ( (void**) &psixH.z.f, size_psix );
	hipMalloc ( (void**) &psixH.z.b, size_psix );

	hipMalloc ( (void**) &psiyE.z.f, size_psiy );
	hipMalloc ( (void**) &psiyE.z.b, size_psiy );
	hipMalloc ( (void**) &psiyE.x.f, size_psiy );
	hipMalloc ( (void**) &psiyE.x.b, size_psiy );
	hipMalloc ( (void**) &psiyH.z.f, size_psiy );
	hipMalloc ( (void**) &psiyH.z.b, size_psiy );
	hipMalloc ( (void**) &psiyH.x.f, size_psiy );
	hipMalloc ( (void**) &psiyH.x.b, size_psiy );

	hipMalloc ( (void**) &psizE.x.f, size_psiz );
	hipMalloc ( (void**) &psizE.x.b, size_psiz );
	hipMalloc ( (void**) &psizE.y.f, size_psiz );
	hipMalloc ( (void**) &psizE.y.b, size_psiz );
	hipMalloc ( (void**) &psizH.x.f, size_psiz );
	hipMalloc ( (void**) &psizH.x.b, size_psiz );
	hipMalloc ( (void**) &psizH.y.f, size_psiz );
	hipMalloc ( (void**) &psizH.y.b, size_psiz );
	
	// --------------------------------------------------------------------------------
	// Initialize the device arrays
	initMainArrays ( (N.x+2)*N.y*N.z, devE );
	initMainArrays ( (N.x+2)*N.y*N.z, devH );
	//initMainArrays ( (N.x+1)*N.y*N.z, devCE );
	
	initPsiArrays ( Ntotpmlx, Db_pmlx, Dg_pmlx, psixE.y, psixE.z );
	initPsiArrays ( Ntotpmly, Db_pmly, Dg_pmly, psiyE.z, psiyE.x );
	initPsiArrays ( Ntotpmlz, Db_pmlz, Dg_pmlz, psizE.x, psizE.y );
	initPsiArrays ( Ntotpmlx, Db_pmlx, Dg_pmlx, psixH.y, psixH.z );
	initPsiArrays ( Ntotpmly, Db_pmly, Dg_pmly, psiyH.z, psiyH.x );
	initPsiArrays ( Ntotpmlz, Db_pmlz, Dg_pmlz, psizH.x, psizH.y );
	
	// --------------------------------------------------------------------------------
	// Copy arrays from host to device
	hipMemcpy ( devCE.x, CE.x[0][0], size_devC, hipMemcpyHostToDevice );
	hipMemcpy ( devCE.y, CE.y[0][0], size_devC, hipMemcpyHostToDevice );
	hipMemcpy ( devCE.z, CE.z[0][0], size_devC, hipMemcpyHostToDevice );

	free(CE.x);
	free(CE.y);
	free(CE.z);

	// --------------------------------------------------------------------------------
	// time loop
	t0 = time(0);
	for ( tstep=1; tstep<=TMAX; tstep++) {
		// E-fields main region update
		for ( i=0; i<NK; i++) updateE <<<Dg_main[i],Db_main,Ns_main>>> ( N, devE, devH, devCE, idx0[i] );
		
		// E-fields CPML region update
		updateCPMLxE <<<Dg_pmlx,Db_pmlx>>> ( N, devE, devH, devCE, psixE.y.f, psixE.z.f, 0);
		updateCPMLxE <<<Dg_pmlx,Db_pmlx>>> ( N, devE, devH, devCE, psixE.y.b, psixE.z.b, 1); 
		updateCPMLyE <<<Dg_pmly,Db_pmly>>> ( N, devE, devH, devCE, psiyE.z.f, psiyE.x.f, 0);
		updateCPMLyE <<<Dg_pmly,Db_pmly>>> ( N, devE, devH, devCE, psiyE.z.b, psiyE.x.b, 1);
		updateCPMLzE <<<Dg_pmlz,Db_pmlz,Ns_pmlz>>> ( N, devE, devH, devCE, psizE.x.f, psizE.y.f, 0);
		updateCPMLzE <<<Dg_pmlz,Db_pmlz,Ns_pmlz>>> ( N, devE, devH, devCE, psizE.x.b, psizE.y.b, 1);
			
		// Source update
		updateSrc <<<DGsrc,DBsrc>>> ( N, devE, tstep );

		// H-fields main region update
		for ( i=0; i<NK; i++) updateH <<<Dg_main[i],Db_main,Ns_main>>> ( N, devE, devH, idx0[i] );
		
		// H-fields CPML region update
		updateCPMLxH <<<Dg_pmlx,Db_pmlx>>> ( N, devE, devH, psixH.y.f, psixH.z.f, 0); 
		updateCPMLxH <<<Dg_pmlx,Db_pmlx>>> ( N, devE, devH, psixH.y.b, psixH.z.b, 1); 
		updateCPMLyH <<<Dg_pmlx,Db_pmlx>>> ( N, devE, devH, psiyH.z.f, psiyH.x.f, 0);
		updateCPMLyH <<<Dg_pmlx,Db_pmlx>>> ( N, devE, devH, psiyH.z.b, psiyH.x.b, 1);
		updateCPMLzH <<<Dg_pmlz,Db_pmlz,Ns_pmlz>>> ( N, devE, devH, psizH.x.f, psizH.y.f, 0);
		updateCPMLzH <<<Dg_pmlz,Db_pmlz,Ns_pmlz>>> ( N, devE, devH, psizH.x.b, psizH.y.b, 1);
			
		if ( tstep/50*50 == tstep ) {
			// Copy arrays from device to host
			hipMemcpy( Ex[0][0], devE.x, (N.x+2)*N.y*N.z*sizeof(float), hipMemcpyDeviceToHost );
			//hipMemcpy( Ez[0][0], devE.z, (N.x+2)*N.y*N.z*sizeof(float), hipMemcpyDeviceToHost );
			//hipMemcpy( CEx[0][0], devCE.x, (N.x+1)*N.y*N.z*sizeof(float), hipMemcpyDeviceToHost );
			hipMemcpy( Ey[0][0], devE.y, (N.x+2)*N.y*N.z*sizeof(float), hipMemcpyDeviceToHost );
			hipMemcpy( Ez[0][0], devE.z, (N.x+2)*N.y*N.z*sizeof(float), hipMemcpyDeviceToHost );
			hipMemcpy( Hx[0][0], devH.x, (N.x+2)*N.y*N.z*sizeof(float), hipMemcpyDeviceToHost );
			hipMemcpy( Hy[0][0], devH.y, (N.x+2)*N.y*N.z*sizeof(float), hipMemcpyDeviceToHost );
			hipMemcpy( Hz[0][0], devH.z, (N.x+2)*N.y*N.z*sizeof(float), hipMemcpyDeviceToHost );

			dumpToH5(N.x+2, N.y, N.z, N.x/2, 0, 0, N.x/2, N.y-1, N.z-1, Ex, "gpu_png/Ex-%05d.h5", tstep);
			exec("h5topng -ZM0.1 -x0 -S4 -c /usr/share/h5utils/colormaps/dkbluered gpu_png/Ex-%05d.h5", tstep);
			dumpToH5(N.x+2, N.y, N.z, N.x/2, 0, 1, N.x/2, N.y-1, N.z-2, Ex, "gpu_png/Ex-%05d.1.h5", tstep);
			exec("h5topng -ZM0.1 -x0 -S4 -c /usr/share/h5utils/colormaps/dkbluered gpu_png/Ex-%05d.1.h5", tstep);
			dumpToH5(N.x+2, N.y, N.z, N.x/2, 0, 2, N.x/2, N.y-1, N.z-3, Ex, "gpu_png/Ex-%05d.2.h5", tstep);
			exec("h5topng -ZM0.1 -x0 -S4 -c /usr/share/h5utils/colormaps/dkbluered gpu_png/Ex-%05d.2.h5", tstep);

			dumpToH5(N.x+2, N.y, N.z, N.x/2, 0, 0, N.x/2, N.y-1, N.z-1, Ey, "gpu_png/Ey-%05d.h5", tstep);
			exec("h5topng -ZM0.1 -x0 -S4 -c /usr/share/h5utils/colormaps/dkbluered gpu_png/Ey-%05d.h5", tstep);
			dumpToH5(N.x+2, N.y, N.z, N.x/2, 0, 0, N.x/2, N.y-1, N.z-1, Ez, "gpu_png/Ez-%05d.h5", tstep);
			exec("h5topng -ZM0.1 -x0 -S4 -c /usr/share/h5utils/colormaps/dkbluered gpu_png/Ez-%05d.h5", tstep);
			dumpToH5(N.x+2, N.y, N.z, N.x/2, 0, 0, N.x/2, N.y-1, N.z-1, Hx, "gpu_png/Hx-%05d.h5", tstep);
			exec("h5topng -ZM0.1 -x0 -S4 -c /usr/share/h5utils/colormaps/dkbluered gpu_png/Hx-%05d.h5", tstep);
			dumpToH5(N.x+2, N.y, N.z, N.x/2, 0, 0, N.x/2, N.y-1, N.z-1, Hy, "gpu_png/Hy-%05d.h5", tstep);
			exec("h5topng -ZM0.1 -x0 -S4 -c /usr/share/h5utils/colormaps/dkbluered gpu_png/Hy-%05d.h5", tstep);
			dumpToH5(N.x+2, N.y, N.z, N.x/2, 0, 0, N.x/2, N.y-1, N.z-1, Hz, "gpu_png/Hz-%05d.h5", tstep);
			exec("h5topng -ZM0.1 -x0 -S4 -c /usr/share/h5utils/colormaps/dkbluered gpu_png/Hz-%05d.h5", tstep);
			//dumpToH5(N.x+2, N.y, N.z, 0, 0, 0, N.x, N.y-1, 0, Ez, "gpu_png/Ez-%05d.h5", tstep);
			
			/*
			dumpToH5(N.x+2, N.y, N.z, 0, 0, N.z/2, N.x+1, N.y-1, N.z/2, Ex, "gpu_png/Ex-%05d.h5", tstep);
			exec("h5topng -ZM0.1 -z0 -S4 -c /usr/share/h5utils/colormaps/dkbluered gpu_png/Ex-%05d.h5", tstep);
			dumpToH5(N.x+2, N.y, N.z, 0, 0, N.z/2, N.x+1, N.y-1, N.z/2, Ey, "gpu_png/Ey-%05d.h5", tstep);
			exec("h5topng -ZM0.1 -z0 -S4 -c /usr/share/h5utils/colormaps/dkbluered gpu_png/Ey-%05d.h5", tstep);
			dumpToH5(N.x+2, N.y, N.z, 0, 0, N.z/2, N.x+1, N.y-1, N.z/2, Ez, "gpu_png/Ez-%05d.h5", tstep);
			exec("h5topng -ZM0.1 -z0 -S4 -c /usr/share/h5utils/colormaps/dkbluered gpu_png/Ez-%05d.h5", tstep);
			dumpToH5(N.x+2, N.y, N.z, 0, 0, N.z/2, N.x+1, N.y-1, N.z/2, Hx, "gpu_png/Hx-%05d.h5", tstep);
			exec("h5topng -ZM0.1 -z0 -S4 -c /usr/share/h5utils/colormaps/dkbluered gpu_png/Hx-%05d.h5", tstep);
			dumpToH5(N.x+2, N.y, N.z, 0, 0, N.z/2, N.x+1, N.y-1, N.z/2, Hy, "gpu_png/Hy-%05d.h5", tstep);
			exec("h5topng -ZM0.1 -z0 -S4 -c /usr/share/h5utils/colormaps/dkbluered gpu_png/Hy-%05d.h5", tstep);
			dumpToH5(N.x+2, N.y, N.z, 0, 0, N.z/2, N.x+1, N.y-1, N.z/2, Hz, "gpu_png/Hz-%05d.h5", tstep);
			exec("h5topng -ZM0.1 -z0 -S4 -c /usr/share/h5utils/colormaps/dkbluered gpu_png/Hz-%05d.h5", tstep);
			*/
			
			//dumpToH5(N.x+1, N.y, N.z, N.x/2, 0, 0, N.x/2, N.y-1, N.z-1, CEx, "gpu_png/CEx-%05d.h5", tstep);
			//exec("h5topng -ZM0.1 -x0 -S4 -c /usr/share/h5utils/colormaps/dkbluered gpu_png/CEx-%05d.h5", tstep);

			updateTimer(t0, tstep, time_str);
			printf("tstep=%d\t%s\n", tstep, time_str);
		}
	}
	updateTimer(t0, tstep, time_str);
	printf("tstep=%d\t%s\n", tstep, time_str);
	
	free(Ex);
	free(Ey);
	free(Ez);
	free(Hx);
	free(Hy);
	free(Hz);
	freeMainArrays ( devE );
	freeMainArrays ( devH );
	freeMainArrays ( devCE );
	freePsiArrays ( psixE, psiyE, psizE );
	freePsiArrays ( psixH, psiyH, psizH );
}
