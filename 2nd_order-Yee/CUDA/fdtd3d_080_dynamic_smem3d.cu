#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdarg.h>
#include <math.h>
#include <hdf5.h>


typedef struct N3 {
	int x, y, z;
} N3;


typedef struct P3F3 {
	float ***x, ***y, ***z;
} P3F3;


typedef struct P1F3 {
	float *x, *y, *z;
} P1F3;


__host__ void updateTimer(time_t t0, int tstep, char str[]) {
	int elapsedTime=(int)(time(0)-t0);
	sprintf(str, "%02d:%02d:%02d", elapsedTime/3600, elapsedTime%3600/60, elapsedTime%60);
}


__host__ void exec(char *format, ...) {
	char str[1024];
	va_list ap;
	va_start(ap, format);
	vsprintf(str, format, ap);
	system(str);
}


__host__ void dumpToH5(int Ni, int Nj, int Nk, int is, int js, int ks, int ie, int je, int ke, float ***f, char *format, ...) {
	char filename[1024];
	va_list ap;
	va_start(ap, format);
	vsprintf(filename, format, ap);
	hid_t file, dataset, filespace, memspace;

	hsize_t dimsm[3] = { Ni, Nj, Nk };
	hsize_t start[3] = { is, js, ks };
	hsize_t count[3] = { 1-is+ie, 1-js+je, 1-ks+ke };
	memspace = H5Screate_simple(3, dimsm, 0);
	filespace = H5Screate_simple(3, count, 0);
	file = H5Fcreate(filename, H5F_ACC_TRUNC, H5P_DEFAULT, H5P_DEFAULT);
	dataset = H5Dcreate(file, "Data", H5T_NATIVE_FLOAT, filespace, H5P_DEFAULT);
	H5Sselect_hyperslab(memspace, H5S_SELECT_SET, start, 0, count, 0);
	H5Dwrite(dataset, H5T_NATIVE_FLOAT, memspace, filespace, H5P_DEFAULT, f[0][0]);
	H5Dclose(dataset);
	H5Sclose(filespace);
	H5Sclose(memspace);
	H5Fclose(file);
}


__host__ void print_array(N3 N, float ***a) {
	int j,k;
	for (j=0; j<N.y; j++) {
		for (k=0; k<N.z; k++) {
			printf("%1.4f\t", a[N.x/2][j][k]);
		}
		printf("\n");
	}
	printf("\n");
}


__host__ float ***makeArray(N3 N) {
	float ***f;

	f = (float ***) calloc (N.x, sizeof(float **));
	f[0] = (float **) calloc (N.y*N.x, sizeof(float *));
	f[0][0] = (float *) calloc (N.z*N.y*N.x, sizeof(float));

	for (int i=0; i<N.x; i++) f[i] = f[0] + i*N.y;
	for (int i=0; i<N.y*N.x; i++) f[0][i] = f[0][0] + i*N.z;

	return f;
}


__host__ void set_geometry(N3 N, P3F3 CE) { 
	int i,j,k;

	for (i=0; i<N.x; i++) {
		for (j=0; j<N.y; j++) {
			for (k=0; k<N.z; k++) {
				CE.x[i][j][k] = 0.5;
				CE.y[i][j][k] = 0.5;
				CE.z[i][j][k] = 0.5;
			}
		}
	}
}


__global__ void initArrays(N3 N, int Nzpit, P1F3 E, P1F3 H) {
	int idx;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	
	//printf("gridDim.x=%d\n",gridDim.x);
	//printf("blockIdx.x=%d, blockDim.x=%d, threadIdx.x=%d\n", blockIdx.x, blockDim.x, threadIdx.x);
	if ( idx < N.x*N.y*Nzpit ) {
		E.x[idx] = 0;
		E.y[idx] = 0;
		E.z[idx] = 0;
		H.x[idx] = 0;
		H.y[idx] = 0;
		H.z[idx] = 0;
	}
}


__global__ void updateE(N3 N, int Nzpit, N3 Dg, N3 Db, P1F3 E, P1F3 H, P1F3 CE) {
	int bk;
	int tk, tj, ti;
	int i, j, k;
	bk = blockIdx.x;
	tk = threadIdx.x;
	tj = threadIdx.y;
	ti = threadIdx.z;
	k = Db.x*( bk%Dg.x ) + tk;
	j = Db.y*( (bk/Dg.x)%Dg.y ) + tj;
	i = Db.z*( bk/(Dg.x*Dg.y) ) + ti;

	//printf("bk(%d),\tbk%Dg.x (%d),\t(bk/Dg.x)%Dg.y (%d),\tbk/(Dg.x*Dg.y) (%d)\n", bk, bk%Dg.x, (bk/Dg.x)%Dg.y, bk/(Dg.x*Dg.y) );
	//printf("blockIdx(%d),\tthreadIdx(%d,%d,%d),\tkji(%d,%d,%d)\n", bk, tk, tj, ti, k, j, i);

	int Bx = Db.x+1;
	int By = Db.x+1;
	int Bz = Db.x;
	int BPx = (Db.y+1)*(Db.x+1);
	int BPy = (Db.y)*(Db.x+1);
	int BPz = (Db.y+1)*(Db.x);
	int Nyzpit = N.y*Nzpit;
	int idx = k + Nzpit*j + Nyzpit*i;
	
	extern __shared__ float hs[];
	float* hx = (float*)hs;
	float* hy = (float*)&hx[(Db.z)*BPx];
	float* hz = (float*)&hy[(Db.z+1)*BPy];

	if ( i<N.x && j<N.y && k<N.z ) {
		//printf("(%d),\t(%d,%d,%d),\t(%d,%d,%d),\t%d\n", bk, tk, tj, ti, k, j, i, idx);

		//__shared__ float hx[(TPBz)*(TPBy+1)*(TPBx+1)];
		//__shared__ float hy[(TPBz+1)*(TPBy)*(TPBx+1)];
		//__shared__ float hz[(TPBz+1)*(TPBy+1)*(TPBx)];

		hx[tk + Bx*tj + BPx*ti] = H.x[idx];
		hy[tk + By*tj + BPy*ti] = H.y[idx];
		hz[tk + Bz*tj + BPz*ti] = H.z[idx];

		if ( tk==Db.x-1 && k<N.z-1 ) {
			hx[(tk+1) + Bx*tj + BPx*ti] = H.x[idx+1];
			hy[(tk+1) + By*tj + BPy*ti] = H.y[idx+1];
		}
		if ( tj==Db.y-1 && j<N.y-1 ) {
			hx[tk + Bx*(tj+1) + BPx*ti] = H.x[idx+Nzpit];
			hz[tk + Bz*(tj+1) + BPz*ti] = H.z[idx+Nzpit];
		}
		if ( ti==Db.z-1 && i<N.x-1 ) {
			hy[tk + By*tj + BPy*(ti+1)] = H.y[idx+Nyzpit];
			hz[tk + Bz*tj + BPz*(ti+1)] = H.z[idx+Nyzpit];
		}
	}
	__syncthreads();

	if ( i<N.x && j<N.y && k<N.z ) {
		if ( j<N.y-1 && k<N.z-1 ) 
			E.x[idx] += CE.x[idx]*( 
					  hz[tk + Bz*(tj+1) + BPz*ti] 
					- hz[tk + Bz*tj + BPz*ti] 
					- hy[(tk+1) + By*tj + BPy*ti]
				   	+ hy[tk + By*tj + BPy*ti] );
		if ( i<N.x-1 && k<N.z-1 ) 
			E.y[idx] += CE.y[idx]*( 
					  hx[(tk+1) + Bx*tj + BPx*ti] 
					- hx[tk + Bx*tj + BPx*ti] 
					- hz[tk + Bz*tj + BPz*(ti+1)] 
					+ hz[tk + Bz*tj + BPz*ti] );
		if ( i<N.x-1 && j<N.y-1 ) 
			E.z[idx] += CE.z[idx]*( 
					  hy[tk + By*tj + BPy*(ti+1)] 
					- hy[tk + By*tj + BPy*ti] 
					- hx[tk + Bx*(tj+1) + BPx*ti] 
					+ hx[tk + Bx*tj + BPx*ti] );
	}
}


__global__ void updateSrc(N3 N, int Nzpit, P1F3 E, int tstep) {
	int idx, ijk;

	idx = blockIdx.x*blockDim.x + threadIdx.x;
	ijk = idx*(N.y)*(Nzpit) + (N.y/2)*(Nzpit) + (N.z/2);

	//printf("idx=%d, ijk=%d\n", idx, ijk);
	//E.x[ijk] += __sinf(0.1*tstep);
	if ( idx < N.x ) {
		E.x[ijk] += sin(0.1*tstep);
	}
}


__global__ void updateH(N3 N, int Nzpit, N3 Dg, N3 Db, P1F3 E, P1F3 H) {
	int bk;
	int tk, tj, ti;
	int i, j, k;
	bk = blockIdx.x;
	tk = threadIdx.x;
	tj = threadIdx.y;
	ti = threadIdx.z;
	k = Db.x*( bk%Dg.x ) + tk;
	j = Db.y*( (bk/Dg.x)%Dg.y ) + tj;
	i = Db.z*( bk/(Dg.x*Dg.y) ) + ti;

	int Bx = Db.x+1;
	int By = Db.x+1;
	int Bz = Db.x;
	int BPx = (Db.y+1)*(Db.x+1);
	int BPy = (Db.y)*(Db.x+1);
	int BPz = (Db.y+1)*(Db.x);
	int Nyzpit = N.y*Nzpit;
	int idx = k + Nzpit*j + Nyzpit*i;
	
	extern __shared__ float es[];
	float* ex = (float*)es;
	float* ey = (float*)&ex[(Db.z)*BPx];
	float* ez = (float*)&ey[(Db.z+1)*BPy];

	if ( i<N.x && j<N.y && k<N.z ) {
		ex[(tk+1) + Bx*(tj+1) + BPx*ti] = E.x[idx];
		ey[(tk+1) + By*tj + BPy*(ti+1)] = E.y[idx];
		ez[tk + Bz*(tj+1) + BPz*(ti+1)] = E.z[idx];
		if ( tk==0 && k>0 ) {
			ex[Bx*(tj+1) + BPx*ti] = E.x[idx-1];
			ey[By*tj + BPy*(ti+1)] = E.y[idx-1];
		}
		if ( tj==0 && j>0 ) {
			ex[(tk+1) + BPx*ti] = E.x[idx-Nzpit];
			ez[tk + BPz*(ti+1)] = E.z[idx-Nzpit];
		}
		if ( ti==0 && i>0 ) {
			ey[(tk+1) + By*tj] = E.y[idx-Nyzpit];
			ez[tk + Bz*(tj+1)] = E.z[idx-Nyzpit];
		}
	}
	__syncthreads();

	if ( i<N.x && j<N.y && k<N.z ) {
		if ( j>0 && k>0 ) 
			H.x[idx] -= 0.5*( 
					  ez[tk + Bz*(tj+1) + BPz*(ti+1)] 
					- ez[tk + Bz*tj + BPz*(ti+1)] 
					- ey[(tk+1) + By*tj + BPy*(ti+1)]
					+ ey[tk + By*tj + BPy*(ti+1)] );
		if ( i>0 && k>0 ) 
			H.y[idx] -= 0.5*( 
					  ex[(tk+1) + Bx*(tj+1) + BPx*ti] 
					- ex[tk + Bx*(tj+1) + BPx*ti] 
					- ez[tk + Bz*(tj+1) + BPz*(ti+1)] 
					+ ez[tk + Bz*(tj+1) + BPz*ti] );
		if ( i>0 && j>0 ) 
			H.z[idx] -= 0.5*( 
					  ey[(tk+1) + By*tj + BPy*(ti+1)] 
					- ey[(tk+1) + By*tj + BPy*ti] 
					- ex[(tk+1) + Bx*(tj+1) + BPx*ti] 
					+ ex[(tk+1) + Bx*tj + BPx*ti] );
	}
}


int main() {
	int tstep;
	char time_str[32];
	time_t t0;

	// Set the parameters
	N3 N;
	N.x = 100;
	N.y = 200;
	N.z = 500;
	//N.y = 16;
	//N.z = 20;
	int TMAX = 1000;
	printf("N(%d,%d,%d), TMAX=%d\n", N.x, N.y, N.z, TMAX);

	// Allocate host memory
	float ***Ex;
	P3F3 CE;
	Ex = makeArray(N);
	CE.x = makeArray(N);
	CE.y = makeArray(N);
	CE.z = makeArray(N);

	// Geometry
	set_geometry(N, CE);


	// Allocate device memory
	P1F3 devE;
	P1F3 devH;
	P1F3 devCE;
	int z_size = N.z*sizeof(float);
	size_t pitch;
	hipMallocPitch ( (void**) &devE.x, &pitch, z_size, N.x*N.y );
	hipMallocPitch ( (void**) &devE.y, &pitch, z_size, N.x*N.y );
	hipMallocPitch ( (void**) &devE.z, &pitch, z_size, N.x*N.y );
	hipMallocPitch ( (void**) &devH.x, &pitch, z_size, N.x*N.y );
	hipMallocPitch ( (void**) &devH.y, &pitch, z_size, N.x*N.y );
	hipMallocPitch ( (void**) &devH.z, &pitch, z_size, N.x*N.y );
	hipMallocPitch ( (void**) &devCE.x, &pitch, z_size, N.x*N.y );
	hipMallocPitch ( (void**) &devCE.y, &pitch, z_size, N.x*N.y );
	hipMallocPitch ( (void**) &devCE.z, &pitch, z_size, N.x*N.y );
	
	// Copy arrays from host to device
	hipMemcpy2D ( devCE.x, pitch, CE.x[0][0], z_size, z_size, N.x*N.y, hipMemcpyHostToDevice );
	hipMemcpy2D ( devCE.y, pitch, CE.y[0][0], z_size, z_size, N.x*N.y, hipMemcpyHostToDevice );
	hipMemcpy2D ( devCE.z, pitch, CE.z[0][0], z_size, z_size, N.x*N.y, hipMemcpyHostToDevice );
	
	int Nz_pitch = pitch/4;
	printf("pitch= %u, Nz_pitch= %d\n", pitch, Nz_pitch);

	// Set the GPU parameters
	N3 TPB;	// Number of threads per block
	TPB.x = 16;
	TPB.y = 4;
	TPB.z = 4;

	N3 BPG;	// Number of thread blocks per grid
	BPG.x = Nz_pitch/TPB.x;
	BPG.y = N.y%TPB.y == 0 ? N.y/TPB.y : N.y/TPB.y + 1;
	BPG.z = N.x%TPB.z == 0 ? N.x/TPB.z : N.x/TPB.z + 1;
	dim3 Dg = dim3(BPG.x*BPG.y*BPG.z);
	dim3 Db = dim3(TPB.x, TPB.y, TPB.z);
	//dim3 Dg = dim3(20);
	//dim3 Db = dim3(16,3,4);
	size_t Ns = sizeof(float)*( 
				(TPB.z)*(TPB.y+1)*(TPB.x+1) +
	            (TPB.z+1)*(TPB.y)*(TPB.x+1) +
	            (TPB.z+1)*(TPB.y+1)*(TPB.x) ); 
	printf("Threads per block: %d (%d,%d,%d)\n", TPB.x*TPB.y*TPB.z, TPB.x, TPB.y, TPB.z);
	if ( TPB.x*TPB.y*TPB.z > 512 ) {
		printf("Error: An excessive number of threads per block.\n");
		exit(0);
	}
	printf("Blocks per grid: %d (%d,%d,%d)\n", BPG.x*BPG.y*BPG.z, BPG.x, BPG.y, BPG.z);
	if ( BPG.x*BPG.y*BPG.z > 65535 ) {
		printf("Error: An excessive number of blocks per grid.\n");
		exit(0);
	}
	printf("Number of bytes in shared memory: %d\n", Ns);

	int TPBsrc = N.x;
	int BPGsrc = 1;
	dim3 Dgsrc(BPGsrc);
	dim3 Dbsrc(TPBsrc);

	int Ntot = N.x*N.y*Nz_pitch;
	int TPBinit = Nz_pitch;
	int BPGinit = Ntot%TPBinit == 0 ? Ntot/TPBinit : Ntot/TPBinit + 1;
	dim3 Dginit(BPGinit);
	dim3 Dbinit(TPBinit);

	// Initialize the device arrays
	initArrays <<<Dginit,Dbinit>>> ( N, Nz_pitch, devE, devH );

	// Main time loop
	t0 = time(0);
	//for ( tstep=1; tstep<=TMAX; tstep++) {
	for ( tstep=1; tstep<=500; tstep++) {
		// Update on the GPU
		updateE <<<Dg,Db,Ns>>> ( N, Nz_pitch, BPG, TPB, devE, devH, devCE );
		updateSrc <<<Dgsrc,Dbsrc>>> ( N, Nz_pitch, devE, tstep );
		updateH <<<Dg,Db,Ns>>> ( N, Nz_pitch, BPG, TPB, devE, devH );

		if ( tstep/10*10 == tstep ) {
			// Copy arrays from device to host
			hipMemcpy2D( Ex[0][0], z_size, devE.x, pitch, z_size, N.x*N.y, hipMemcpyDeviceToHost );

			//print_array(N, Ex);
			dumpToH5(N.x, N.y, N.z, N.x/2, 0, 0, N.x/2, N.y-1, N.z-1, Ex, "gpu_png/Ex-%05d.h5", tstep);
			exec("h5topng -ZM0.1 -x0 -S4 -c /usr/share/h5utils/colormaps/dkbluered gpu_png/Ex-%05d.h5", tstep);

			updateTimer(t0, tstep, time_str);
			printf("tstep=%d\t%s\n", tstep, time_str);
		}
	}
	updateTimer(t0, tstep, time_str);
	printf("tstep=%d\t%s\n", tstep, time_str);
}
