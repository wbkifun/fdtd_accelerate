#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdarg.h>
#include <math.h>
#include <hdf5.h>

#define Npml 10

const float light_velocity = 2.99792458e8;	// m s- 
const float ep0 = 8.85418781762038920e-12;	// F m-1 (permittivity at vacuum)
const float	mu0 = 1.25663706143591730e-6;	// N A-2 (permeability at vacuum)
const float imp0 = sqrt( mu0/ep0 );	// (impedance at vacuum)
const float pi = 3.14159265358979323846;

const int MAX_BPG = 65535;

// Allocate constant memory for CPML
__constant__ float rcmbE[2*Npml];
__constant__ float rcmaE[2*Npml];
__constant__ float rcmbH[2*Npml];
__constant__ float rcmaH[2*Npml];


typedef struct N3 {
	int x, y, z;
} N3;


typedef struct N3dim3 {
	dim3 x, y, z;
} N3dim3;


typedef struct P3F3 {
	float ***x, ***y, ***z;
} P3F3;


typedef struct P1F3 {
	float *x, *y, *z;
} P1F3;


typedef struct P1F2 {
	float *f, *b;
} P1F2;


typedef struct P1F6 {
	P1F2 x, y, z;
} P1F6;


__host__ void updateTimer(time_t t0, int tstep, char str[]) {
	int elapsedTime=(int)(time(0)-t0);
	sprintf(str, "%02d:%02d:%02d", elapsedTime/3600, elapsedTime%3600/60, elapsedTime%60);
}


__host__ void exec(char *format, ...) {
	char str[1024];
	va_list ap;
	va_start(ap, format);
	vsprintf(str, format, ap);
	system(str);
}


__host__ void dumpToH5(int Ni, int Nj, int Nk, int is, int js, int ks, int ie, int je, int ke, float ***f, char *format, ...) {
	char filename[1024];
	va_list ap;
	va_start(ap, format);
	vsprintf(filename, format, ap);
	hid_t file, dataset, filespace, memspace;

	hsize_t dimsm[3] = { Ni, Nj, Nk };
	hsize_t start[3] = { is, js, ks };
	hsize_t count[3] = { 1-is+ie, 1-js+je, 1-ks+ke };
	memspace = H5Screate_simple(3, dimsm, 0);
	filespace = H5Screate_simple(3, count, 0);
	file = H5Fcreate(filename, H5F_ACC_TRUNC, H5P_DEFAULT, H5P_DEFAULT);
	dataset = H5Dcreate(file, "Data", H5T_NATIVE_FLOAT, filespace, H5P_DEFAULT);
	H5Sselect_hyperslab(memspace, H5S_SELECT_SET, start, 0, count, 0);
	H5Dwrite(dataset, H5T_NATIVE_FLOAT, memspace, filespace, H5P_DEFAULT, f[0][0]);
	H5Dclose(dataset);
	H5Sclose(filespace);
	H5Sclose(memspace);
	H5Fclose(file);
}


__host__ void print_array(N3 N, float ***a) {
	int j,k;
	for (j=0; j<N.y; j++) {
		for (k=0; k<N.z; k++) {
			printf("%1.4f\t", a[N.x/2][j][k]);
		}
		printf("\n");
	}
	printf("\n");
}


__host__ float ***makeArray(N3 N) {
	float ***f;

	f = (float ***) calloc (N.x, sizeof(float **));
	f[0] = (float **) calloc (N.y*N.x, sizeof(float *));
	f[0][0] = (float *) calloc (N.z*N.y*N.x, sizeof(float));

	for (int i=0; i<N.x; i++) f[i] = f[0] + i*N.y;
	for (int i=0; i<N.y*N.x; i++) f[0][i] = f[0][0] + i*N.z;

	return f;
}


__host__ void set_geometry(N3 N, P3F3 CE) {
	int i,j,k;

	for (i=0; i<N.x; i++) {
		for (j=0; j<N.y; j++) {
			for (k=0; k<N.z; k++) {
				CE.x[i][j][k] = 0.5;
				CE.y[i][j][k] = 0.5;
				CE.z[i][j][k] = 0.5;
			}
		}
	}
}


__host__ void verify_16xNz(int Nz) {
	int R = Nz%16;
	int N1 = Nz-R; 
	int N2 = N1+16; 
	if ( R == 0 ) printf("Nz is a multiple of 16.\n");
	else {
		printf("Error: Nz is not a multiple of 16.\n");
		printf("Recommend Nz: %d or %d\n", N1, N2);
		exit(0);
	}
}


__host__ int selectTPB(int Ntot) {
	float occupancy;
	float max_occupancy=0;
	int selTPB=0;
	int TPB;	// thread/block
	int WPB;	// wrap/block
	int ABPM;	// active block/streaming multiprocessor
	int AWPM;	// active warp/streaming multiprocessor
	int MAX_ABPM = 8;	 
	int MAX_AWPM = 32;	 
	int MAX_TPB = 512;	 
	//int MAX_TPM = 1024;	 
	int TPW = 32;	// thread/warp

	for ( TPB=MAX_TPB; TPB>0; TPB-- ) {
		if ( Ntot%TPB == 0 && TPB%16 == 0 ) {
			WPB = TPB%TPW == 0 ? TPB/TPW : TPB/TPW+1;
			ABPM = MAX_AWPM/WPB <= 8 ? MAX_AWPM/WPB : MAX_ABPM;
			AWPM = WPB*ABPM;
			occupancy = (float)AWPM/MAX_AWPM;
			if ( max_occupancy < occupancy ) {
				max_occupancy = occupancy;
				selTPB = TPB;
			}
			//printf("TPB=%d, WPB=%d, ABPM=%d, AWPM=%d, occupancy=%g, max_occupancy=%g, selTPB=%d\n", TPB, WPB, ABPM, AWPM, occupancy, max_occupancy, selTPB);
		}
	}

	if ( selTPB == 0 ) {
		printf("Error: There is not a TPB which is a aliquot part of the Ntot(%d).\n", Ntot);
		exit(0);
	}

	printf("Occupancy=%1.2f\n", max_occupancy);
	return selTPB;
}


__global__ void initArray(int Ntot, float *a, int idx0) {
	int idx = idx0 + blockIdx.x*blockDim.x + threadIdx.x;

	if ( idx < Ntot ) a[idx] = 0;
}


__host__ void initMainArrays(N3 N, P1F3 F) {
	int Ntot = (N.x+1)*N.y*N.z;
	int TPB = selectTPB( Ntot );
	int BPG = Ntot/TPB;

	int Nkernel = BPG/MAX_BPG + 1;
	int sBPG = BPG/Nkernel;
	int i, idx0;
	for ( i=0; i<Nkernel-1; i++) {
		idx0 = TPB*sBPG*i;
		initArray <<<dim3(sBPG),dim3(TPB)>>> (Ntot, F.x, idx0); 
		initArray <<<dim3(sBPG),dim3(TPB)>>> (Ntot, F.y, idx0); 
		initArray <<<dim3(sBPG),dim3(TPB)>>> (Ntot, F.z, idx0); 
	}
	idx0 = TPB*sBPG*(Nkernel-1);
	initArray <<<dim3(sBPG+BPG%Nkernel),dim3(TPB)>>> (Ntot, F.x, idx0); 
	initArray <<<dim3(sBPG+BPG%Nkernel),dim3(TPB)>>> (Ntot, F.y, idx0); 
	initArray <<<dim3(sBPG+BPG%Nkernel),dim3(TPB)>>> (Ntot, F.z, idx0); 

	printf("main init: Ntot=%d(%dx%dx%d), TPB=%d, BPG=%d, sBPG(%d)=%d\n", Ntot, N.x+1, N.y, N.z, TPB, BPG, Nkernel, sBPG);
}


__host__ void initPsiArrays(N3 N, N3 Ntot, N3dim3 DGpml, N3dim3 DBpml, P1F6 psix, P1F6 psiy, P1F6 psiz) {
	initArray <<<DGpml.x,DBpml.x>>> (Ntot.x, psix.y.f, 0); 
	initArray <<<DGpml.x,DBpml.x>>> (Ntot.x, psix.y.b, 0); 
	initArray <<<DGpml.x,DBpml.x>>> (Ntot.x, psix.z.f, 0); 
	initArray <<<DGpml.x,DBpml.x>>> (Ntot.x, psix.z.b, 0); 
	
	initArray <<<DGpml.y,DBpml.y>>> (Ntot.y, psiy.z.f, 0); 
	initArray <<<DGpml.y,DBpml.y>>> (Ntot.y, psiy.z.b, 0); 
	initArray <<<DGpml.y,DBpml.y>>> (Ntot.y, psiy.x.f, 0); 
	initArray <<<DGpml.y,DBpml.y>>> (Ntot.y, psiy.x.b, 0); 

	initArray <<<DGpml.z,DBpml.z>>> (Ntot.y, psiz.x.f, 0); 
	initArray <<<DGpml.z,DBpml.z>>> (Ntot.y, psiz.x.b, 0); 
	initArray <<<DGpml.z,DBpml.z>>> (Ntot.y, psiz.y.f, 0); 
	initArray <<<DGpml.z,DBpml.z>>> (Ntot.y, psiz.y.b, 0); 
}


__host__ void freeMainArrays(P1F3 F) {
	hipFree(F.x);
	hipFree(F.y);
	hipFree(F.z);
}


__host__ void freePsiArrays(P1F6 psix, P1F6 psiy, P1F6 psiz) {
	hipFree(psix.y.f);
	hipFree(psix.y.b);
	hipFree(psix.z.f);
	hipFree(psix.z.b);

	hipFree(psiy.z.f);
	hipFree(psiy.z.b);
	hipFree(psiy.x.f);
	hipFree(psiy.x.b);

	hipFree(psiz.x.f);
	hipFree(psiz.x.b);
	hipFree(psiz.y.f);
	hipFree(psiz.y.b);
}


__global__ void updateE(N3 N, int TPB, P1F3 E, P1F3 H, P1F3 CE) {
	int tk = threadIdx.x;
	int idx = blockIdx.x*TPB + tk;
	int Nyz = N.y*N.z;
	int eidx = idx + Nyz;

	extern __shared__ float hs[];
	float* hx = (float*) hs;
	float* hy = (float*) &hx[TPB+1];
	float* hz = (float*) &hy[TPB+1];

	hx[tk] = H.x[idx];
	hy[tk] = H.y[idx];
	hz[tk] = H.z[idx];
	if ( tk==TPB-1 ) {
		hx[tk+1] = H.x[idx+1];
		hy[tk+1] = H.y[idx+1];
	}
	__syncthreads();

	E.x[eidx] += CE.x[idx]*( H.z[idx+N.z] - hz[tk] - hy[tk+1] + hy[tk] );
	E.y[eidx] += CE.y[idx]*( hx[tk+1] - hx[tk] - H.z[idx+Nyz] + hz[tk] );
	E.z[eidx] += CE.z[idx]*( H.y[idx+Nyz] - hy[tk] - H.x[idx+N.z] + hx[tk] );
}


__global__ void updateH(N3 N, int TPB, P1F3 E, P1F3 H) {
	int tk = threadIdx.x;
	int idx = blockIdx.x*TPB + tk;
	int Nyz = N.y*N.z;
	int eidx = idx + Nyz;

	extern __shared__ float es[];
	float* ex = (float*) es;
	float* ey = (float*) &ex[TPB+1];
	float* ez = (float*) &ey[TPB+1];

	ex[tk+1] = E.x[eidx];
	ey[tk+1] = E.y[eidx];
	ez[tk] = E.z[eidx];
	if ( tk==0 ) {
		ex[0] = E.x[eidx-1];
		ey[0] = E.y[eidx-1];
	}
	__syncthreads();

	H.x[idx] -= 0.5*( ez[tk] - E.z[eidx-N.z] - ey[tk+1] + ey[tk] );
	H.y[idx] -= 0.5*( ex[tk+1] - ex[tk] - ez[tk] + E.z[eidx-Nyz] );
	H.z[idx] -= 0.5*( ey[tk+1] - E.y[eidx-Nyz] - ex[tk+1] + E.x[eidx-N.z] );
}


__global__ void updateSrc(N3 N, P1F3 E, int tstep) {
	int idx = threadIdx.x;
	//int ijk = idx*N.y*N.z + (N.y/2)*N.z + (N.z/2);
	int ijk = (N.x/2+1)*N.y*N.z + (N.y/2)*N.z + idx;

	//E.x[ijk] += sin(0.1*tstep);
	E.z[ijk] += sin(0.1*tstep);
}


__global__ void updateCPMLxE(N3 N, P1F3 E, P1F3 H, P1F3 CE, float *psi1, float *psi2, int backward) {
	int pidx = blockIdx.x*blockDim.x + threadIdx.x;
	int Nyz = N.y*N.z;
	int pi = pidx/Nyz + backward*Npml;

	int idx = pidx + backward*(N.x-Npml-1)*Nyz;
	int eidx = idx + Nyz;

	psi1[pidx] = rcmbE[pi]*psi1[pidx] + rcmaE[pi]*( H.z[idx+Nyz] - H.z[idx] );
	E.y[eidx] -= CE.y[idx]*psi1[pidx];

	psi2[pidx] = rcmbE[pi]*psi2[pidx] + rcmaE[pi]*( H.y[idx+Nyz] - H.y[idx] );
	E.z[eidx] += CE.z[idx]*psi2[pidx];
}


__global__ void updateCPMLxH(N3 N, P1F3 E, P1F3 H, float *psi1, float *psi2, int backward) {
	int pidx = blockIdx.x*blockDim.x + threadIdx.x;
	int Nyz = N.y*N.z;
	int pi = pidx/Nyz + backward*Npml;

	int idx = pidx + backward*(N.x-Npml)*Nyz;
	int eidx = idx + Nyz;

	psi1[pidx] = rcmbH[pi]*psi1[pidx] + rcmaH[pi]*( E.z[eidx] - E.z[eidx-Nyz] );
	H.y[idx] += 0.5*psi1[pidx];

	psi2[pidx] = rcmbH[pi]*psi2[pidx] + rcmaH[pi]*( E.y[eidx] - E.y[eidx-Nyz] );
	H.z[idx] -= 0.5*psi2[pidx];
}


__global__ void updateCPMLyE(N3 N, P1F3 E, P1F3 H, P1F3 CE, float *psi1, float *psi2, int backward) {
	int pidx = blockIdx.x*blockDim.x + threadIdx.x;
	int i = pidx/(Npml*N.z);
	int pj = ( pidx - i*Npml*N.z )/N.z + backward*Npml;

	int idx = pidx + (i+backward)*(N.y-Npml)*N.z - backward*N.z;
	int eidx = idx + N.y*N.z;

	psi1[pidx] = rcmbE[pj]*psi1[pidx] + rcmaE[pj]*( H.x[idx+N.z] - H.x[idx] );
	E.z[eidx] -= CE.z[idx]*psi1[pidx];

	psi2[pidx] = rcmbE[pj]*psi2[pidx] + rcmaE[pj]*( H.z[idx+N.z] - H.z[idx] );
	E.x[eidx] += CE.x[idx]*psi2[pidx];
}


__global__ void updateCPMLyH(N3 N, P1F3 E, P1F3 H, float *psi1, float *psi2, int backward) {
	int pidx = blockIdx.x*blockDim.x + threadIdx.x;
	int i = pidx/(Npml*N.z);
	int pj = ( pidx - i*Npml*N.z )/N.z + backward*Npml;

	int idx = pidx + (i+backward)*(N.y-Npml)*N.z;
	int eidx = idx + N.y*N.z;

	psi1[pidx] = rcmbH[pj]*psi1[pidx] + rcmaH[pj]*( E.x[eidx] - E.x[eidx-N.z] );
	H.z[idx] += 0.5*psi1[pidx];

	psi2[pidx] = rcmbH[pj]*psi2[pidx] + rcmaH[pj]*( E.z[eidx] - E.z[eidx-N.z] );
	H.x[idx] -= 0.5*psi2[pidx];
}


__global__ void updateCPMLzE(N3 N, P1F3 E, P1F3 H, P1F3 CE, float *psi1, float *psi2, int backward) {
	int tk = threadIdx.x;
	int pidx = blockIdx.x*blockDim.x + tk;
	int Npmlp = Npml+1;
	int i = pidx/(N.y*Npmlp);
	int j = ( pidx - i*N.y*Npmlp )/Npmlp;
	int pk = pidx - i*N.y*Npmlp - j*Npmlp;// + backward*Npml;

	int idx = pidx + (j+i*N.y)*(Nzpit-Npmlp);// + backward*(N.z-Npml-1);
	int eidx = idx + N.y*Nzpit;

	extern __shared__ float hs[];
	float* hx = (float*) hs;
	float* hy = (float*) &hx[TPB+1];

	hx[tk] = H.x[idx];
	hy[tk] = H.y[idx];
	if ( tk==TPB-1 ) {
		hx[tk+1] = H.x[idx+1];
		hy[tk+1] = H.y[idx+1];
	}
	__syncthreads();

	if ( i<N.x-1 && pk<Npml ) {
		psi1[pidx] = rcmbE[pk]*psi1[pidx] + rcmaE[pk]*( hy[tk+1] - hy[tk] );
		E.x[eidx] -= CE.x[idx]*psi1[pidx];

		psi2[pidx] = rcmbE[pk]*psi2[pidx] + rcmaE[pk]*( hx[tk+1] - hx[tk] );
		E.y[eidx] += CE.y[idx]*psi2[pidx];
	}
}


__global__ void updateCPMLzH(
		N3 N, int Nzpit, int TPB,
		P1F3 E, P1F3 H,
		float *psi1, float *psi2,
		int backward) {
	int tk = threadIdx.x;
	int pidx = blockIdx.x*TPB + tk;
	int Npmlp = Npml+1;
	int i = pidx/(N.y*Npmlp);
	int j = ( pidx - i*N.y*Npmlp )/Npmlp;
	int pk = pidx - i*N.y*Npmlp - j*Npmlp;// + backward*Npml;

	int idx = pidx + (j+i*N.y)*(Nzpit-Npmlp);// + backward*(N.z-Npml-1);
	int eidx = idx + N.y*Nzpit;

	extern __shared__ float es[];
	float* ex = (float*) es;
	float* ey = (float*) &ex[TPB+1];

	//printf("[%.3d,%.3d,%.3d]\t\t%d\t\t%d\t\t%d\n",i,j,pk,pidx,idx,eidx);
	if ( i<N.x ) {
		ex[tk+1] = E.x[eidx];
		ey[tk+1] = E.y[eidx];
		if ( tk==0 ) {
			ex[0] = E.x[eidx-1];
			ey[0] = E.y[eidx-1];
		}
	}
	__syncthreads();

	if ( i>0 && i<N.x && pk<Npml ) {
		psi1[pidx] = rcmbH[pk]*psi1[pidx] + rcmaH[pk]*( ey[tk+1] - ey[tk] );
		H.x[idx] += 0.5*psi1[pidx];

		psi2[pidx] = rcmbH[pk]*psi2[pidx] + rcmaH[pk]*( ex[tk+1] - ex[tk] );
		H.y[idx] -= 0.5*psi2[pidx];
	}
}


__global__ void init_boundary_xE(N3 N, P1F3 E) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int eidx = idx + N.x*N.y*N.z;

	//if ( idx/(N.y*N.z) == 0 ) {
		E.y[eidx] = 0;
		E.z[eidx] = 0;
	//}
}


__global__ void init_boundary_yE(N3 N, P1F3 E) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int i = idx/N.z;
	int k = idx%N.z;
	int eidx = (i+1)*N.y*N.z + (N.y-1)*N.z + k;

	//if ( i<N.x ) {
		E.z[eidx] = 0;
		E.x[eidx] = 0;
	//}
}


__global__ void init_boundary_zE(N3 N, P1F3 E) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int i = idx/N.y;
	int j = idx%N.y;
	int eidx = (i+1)*N.y*N.z + j*N.z + (N.z-1);

	//if ( i<N.x ) {
		E.x[eidx] = 0;
		E.y[eidx] = 0;
	//}
}


__global__ void init_boundary_xH(N3 N, P1F3 H) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	//if ( idx/(N.y*N.z) == 0 ) {
		H.x[idx] = 0;
		H.y[idx] = 0;
		H.z[idx] = 0;
	//}
}


__global__ void init_boundary_yH(N3 N, P1F3 H) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int i = idx/N.z;
	int k = idx%N.z;
	int hidx = i*N.y*N.z + k;

	//if ( i<N.x ) {
		H.x[hidx] = 0;
		H.y[hidx] = 0;
		H.z[hidx] = 0;
	//}
}


__global__ void init_boundary_zH(N3 N, P1F3 H) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int i = idx/N.y;
	int j = idx%N.y;
	int hidx = i*N.y*N.z + j*N.z;

	//int i0 = idx/(N.y*N.z);
	//int j0 = ( idx - i*N.y*N.z )/N.z;
	//int k0 = idx%N.z;
	//printf("[%d,%d,%d] %d\n", i0,j0,k0,idx);

	//if ( i<N.x ) {
		//printf("\t\t\tIn: [%d,%d,%d] %d\n", i0,j0,k0,idx);
		H.x[hidx] = 0;
		H.y[hidx] = 0;
		H.z[hidx] = 0;
	//}
}


int main() {
	int tstep;
	char time_str[32];
	time_t t0;
	int i;

	// --------------------------------------------------------------------------------
	// Set the parameters
	N3 N;
	N.x = 400;
	N.y = 200;
	N.z = 208;
	//N.y = 16;
	//N.z = 20;
	verify_16xNz( N.z );
	int TMAX = 10000;
	
	float S = 0.5;
	float dx = 10e-9;
	float dt = S*dx/light_velocity;
	printf("NPML=%d\n", Npml);
	printf("N(%d,%d,%d), TMAX=%d\n", N.x, N.y, N.z, TMAX);

	// --------------------------------------------------------------------------------
	// Allocate host memory
	P3F3 CE;
	CE.x = makeArray(N);
	CE.y = makeArray(N);
	CE.z = makeArray(N);
	float ***Ex, ***Ez;
	N3 Nxp;
	Nxp.x = N.x+1;
	Nxp.y = N.y;
	Nxp.z = N.z;
	Ex = makeArray(Nxp);
	Ez = makeArray(Nxp);

	// --------------------------------------------------------------------------------
	// Geometry
	set_geometry(N, CE);

	// --------------------------------------------------------------------------------
	// Parameters for CPML
	int m = 4;	// grade_order
	float sigma_max = (m+1.)/(15*pi*Npml*dx);
	float alpha = 0.05;
	float *sigmaE, *bE, *aE;
	float *sigmaH, *bH, *aH;

	sigmaE = (float *) calloc (2*Npml, sizeof(float));
	sigmaH = (float *) calloc (2*Npml, sizeof(float));
	bE = (float *) calloc (2*Npml, sizeof(float));
	bH = (float *) calloc (2*Npml, sizeof(float));
	aE = (float *) calloc (2*Npml, sizeof(float));
	aH = (float *) calloc (2*Npml, sizeof(float));
	for (i=0; i<Npml; i++) {
		sigmaE[i] = pow( (Npml-0.5-i)/Npml, m )*sigma_max;
		sigmaE[i+Npml] = pow( (0.5+i)/Npml, m )*sigma_max;
		sigmaH[i] = pow( (float)(Npml-i)/Npml, m )*sigma_max;
		sigmaH[i+Npml] = pow( (1.+i)/Npml, m )*sigma_max;
	}

	for (i=0; i<2*Npml; i++) {
		bE[i] = exp( -(sigmaE[i] + alpha)*dt/ep0 );
		bH[i] = exp( -(sigmaH[i] + alpha)*dt/ep0 );
		aE[i] = sigmaE[i]/(sigmaE[i]+alpha)*(bE[i]-1);
		aH[i] = sigmaH[i]/(sigmaH[i]+alpha)*(bH[i]-1);
		//printf("[%d]\tsigmaE=%g,\tbE=%g,aE=%g\n", i, sigmaE[i], bE[i], aE[i]);
		//printf("[%d]\tsigmaH=%g,\tbH=%g,aH=%g\n", i, sigmaH[i], bH[i], aH[i]);
	}
	free(sigmaE);
	free(sigmaH);

	// --------------------------------------------------------------------------------
	// Copy arrays from host to constant memory
	hipMemcpyToSymbol(HIP_SYMBOL(rcmbE), bE, 2*Npml*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(rcmaE), aE, 2*Npml*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(rcmbH), bH, 2*Npml*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(rcmaH), aH, 2*Npml*sizeof(float));

	// --------------------------------------------------------------------------------
	// Allocate device memory
	P1F3 devE, devH;
	P1F3 devCE;

	hipMalloc ( (void**) &devE.x, (N.x+1)*N.y*N.z*sizeof(float) );
	hipMalloc ( (void**) &devE.y, (N.x+1)*N.y*N.z*sizeof(float) );
	hipMalloc ( (void**) &devE.z, (N.x+1)*N.y*N.z*sizeof(float) );
	hipMalloc ( (void**) &devH.x, (N.x+1)*N.y*N.z*sizeof(float) );
	hipMalloc ( (void**) &devH.y, (N.x+1)*N.y*N.z*sizeof(float) );
	hipMalloc ( (void**) &devH.z, (N.x+1)*N.y*N.z*sizeof(float) );
	hipMalloc ( (void**) &devCE.x, N.x*N.y*N.z*sizeof(float) );
	hipMalloc ( (void**) &devCE.y, N.x*N.y*N.z*sizeof(float) );
	hipMalloc ( (void**) &devCE.z, N.x*N.y*N.z*sizeof(float) );
	
	// Allocate device memory for CPML
	P1F6 psixE, psiyE, psizE;
	P1F6 psixH, psiyH, psizH;

	hipMalloc ( (void**) &psixE.y.f, Npml*N.y*N.z*sizeof(float) );
	hipMalloc ( (void**) &psixE.y.b, Npml*N.y*N.z*sizeof(float) );
	hipMalloc ( (void**) &psixE.z.f, Npml*N.y*N.z*sizeof(float) );
	hipMalloc ( (void**) &psixE.z.b, Npml*N.y*N.z*sizeof(float) );
	hipMalloc ( (void**) &psixH.y.f, Npml*N.y*N.z*sizeof(float) );
	hipMalloc ( (void**) &psixH.y.b, Npml*N.y*N.z*sizeof(float) );
	hipMalloc ( (void**) &psixH.z.f, Npml*N.y*N.z*sizeof(float) );
	hipMalloc ( (void**) &psixH.z.b, Npml*N.y*N.z*sizeof(float) );

	hipMalloc ( (void**) &psiyE.z.f, N.x*Npml*N.z*sizeof(float) );
	hipMalloc ( (void**) &psiyE.z.b, N.x*Npml*N.z*sizeof(float) );
	hipMalloc ( (void**) &psiyE.x.f, N.x*Npml*N.z*sizeof(float) );
	hipMalloc ( (void**) &psiyE.x.b, N.x*Npml*N.z*sizeof(float) );
	hipMalloc ( (void**) &psiyH.z.f, N.x*Npml*N.z*sizeof(float) );
	hipMalloc ( (void**) &psiyH.z.b, N.x*Npml*N.z*sizeof(float) );
	hipMalloc ( (void**) &psiyH.x.f, N.x*Npml*N.z*sizeof(float) );
	hipMalloc ( (void**) &psiyH.x.b, N.x*Npml*N.z*sizeof(float) );

	size_t pml_pitch;
	hipMallocPitch ( (void**) &psizE.x.f, &pml_pitch, Npml*sizeof(float), N.x*N.y );
	hipMallocPitch ( (void**) &psizE.x.b, &pml_pitch, Npml*sizeof(float), N.x*N.y );
	hipMallocPitch ( (void**) &psizE.y.f, &pml_pitch, Npml*sizeof(float), N.x*N.y );
	hipMallocPitch ( (void**) &psizE.y.b, &pml_pitch, Npml*sizeof(float), N.x*N.y );
	hipMallocPitch ( (void**) &psizH.x.f, &pml_pitch, Npml*sizeof(float), N.x*N.y );
	hipMallocPitch ( (void**) &psizH.x.b, &pml_pitch, Npml*sizeof(float), N.x*N.y );
	hipMallocPitch ( (void**) &psizH.y.f, &pml_pitch, Npml*sizeof(float), N.x*N.y );
	hipMallocPitch ( (void**) &psizH.y.b, &pml_pitch, Npml*sizeof(float), N.x*N.y );

	// --------------------------------------------------------------------------------
	// Copy arrays from host to device
	hipMemcpy ( devCE.x, CE.x[0][0], N.x*N.y*N.z*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy ( devCE.y, CE.x[0][0], N.x*N.y*N.z*sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy ( devCE.z, CE.x[0][0], N.x*N.y*N.z*sizeof(float), hipMemcpyHostToDevice );

	free(CE.x);
	free(CE.y);
	free(CE.z);

	// --------------------------------------------------------------------------------
	// Set the GPU parameters
	// TPB: Number of threads per block
	// BPG: Number of thread blocks per grid
	int Ntot, TPB, BPG;

	// main update
	Ntot = N.x*N.y*N.z;
	TPB = selectTPB( Ntot );	 
	//BPG = Ntot%TPB == 0 ? Ntot/TPB : Ntot/TPB + 1;
	BPG = Ntot/TPB;
	dim3 DBmain = dim3(TPB);
	dim3 DGmain = dim3(BPG);
	size_t NSmain = sizeof(float)*( 2*(TPB+1)+TPB );
	printf("main: Ntot=%d(%dx%dx%d), TPB=%d, BPG=%d, NS=%d\n", Ntot, N.x, N.y, N.z, TPB, BPG, NSmain);
	int TPBmain = TPB;	 
	
	// source 
	//TPB = N.x;
	TPB = N.z;
	BPG = 1;
	dim3 DBsrc(TPB);
	dim3 DGsrc(BPG);
	printf("source: TPB=%d, BPG=%d\n", TPB, BPG);

	// init the boundary
	N3dim3 DBinitb, DGinitb;

	Ntot = N.y*N.z;
	TPB = selectTPB( Ntot );	 
	BPG = Ntot/TPB;
	DBinitb.x = dim3(TPB);
	DGinitb.x = dim3(BPG);
	printf("init the boundary (x): Ntot=%d(%dx%d), TPB=%d, BPG=%d\n", Ntot, N.y, N.z, TPB, BPG);

	Ntot = N.x*N.y;
	TPB = selectTPB( Ntot );	 
	BPG = Ntot/TPB;
	DBinitb.y = dim3(TPB);
	DGinitb.y = dim3(BPG);
	printf("init the boundary (y): Ntot=%d(%dx%d), TPB=%d, BPG=%d\n", Ntot, N.x, N.y, TPB, BPG);

	Ntot = N.x*N.y;
	TPB = selectTPB( Ntot );	 
	BPG = Ntot/TPB;
	DBinitb.z = dim3(TPB);
	DGinitb.z = dim3(BPG);
	printf("init the boundary (z): Ntot=%d(%dx%d), TPB=%d, BPG=%d\n", Ntot, N.x, N.y, TPB, BPG);

	// cpml 
	N3 Ntotpml;
	N3dim3 DBpml, DGpml;

	Ntot = Npml*N.y*N.z;
	TPB = selectTPB( Ntot );	 
	BPG = Ntot/TPB;
	DBpml.x = dim3(TPB);
	DGpml.x = dim3(BPG);
	printf("pml (x): Ntot=%d(%dx%dx%d), TPB=%d, BPG=%d\n", Ntot, Npml, N.y, N.z, TPB, BPG);
	Ntotpml.x = Ntot;

	Ntot = N.x*Npml*N.z;
	TPB = selectTPB( Ntot );	 
	BPG = Ntot/TPB;
	DBpml.y = dim3(TPB);
	DGpml.y = dim3(BPG);
	printf("pml (y): Ntot=%d(%dx%dx%d), TPB=%d, BPG=%d\n", Ntot, N.x, Npml, N.z, TPB, BPG);
	Ntotpml.y = Ntot;

	int Npml_pitch = Npml/16 + 16; 
	Ntot = N.x*N.y*Npml_pitch;
	TPB = selectTPB( Ntot );	 
	BPG = Ntot/TPB;
	DBpml.z = dim3(TPB);
	DGpml.z = dim3(BPG);
	printf("pml (z): Ntot=%d(%dx%dx%d), TPB=%d, BPG=%d\n", Ntot, N.x, N.y, Npml_pitch, TPB, BPG);
	Ntotpml.z = Ntot;

	// --------------------------------------------------------------------------------
	// Initialize the device arrays
	initMainArrays ( N, devE );
	initMainArrays ( N, devH );
	initPsiArrays ( N, Ntotpml, DGpml, DBpml, psixE, psiyE, psizE );
	initPsiArrays ( N, Ntotpml, DGpml, DBpml, psixH, psiyH, psizH );

	// --------------------------------------------------------------------------------
	// Main time loop
	t0 = time(0);
	//for ( tstep=1; tstep<=TMAX; tstep++) {
	for ( tstep=1; tstep<=500; tstep++) {
		// Update on the GPU
		updateE <<<DGmain,DBmain,NSmain>>> ( N, TPBmain, devE, devH, devCE );
		//init_boundary_xE <<<DGinitb.x,DBinitb.x>>> ( N, devE );
		//init_boundary_yE <<<DGinitb.y,DBinitb.y>>> ( N, devE );
		init_boundary_zE <<<DGinitb.z,DBinitb.z>>> ( N, devE );
		updateCPMLxE <<<DGpml.x,DBpml.x>>> ( N, devE, devH, devCE, psixE.y.f, psixE.z.f, 0);
		updateCPMLxE <<<DGpml.x,DBpml.x>>> ( N, devE, devH, devCE, psixE.y.b, psixE.z.b, 1); 
		updateCPMLyE <<<DGpml.y,DBpml.y>>> ( N, devE, devH, devCE, psiyE.z.f, psiyE.x.f, 0);
		updateCPMLyE <<<DGpml.y,DBpml.y>>> ( N, devE, devH, devCE, psiyE.z.b, psiyE.x.b, 1);
		//updateCPMLzE <<<DGpmlz,DBpmlz,NSpmlz>>> ( N, Nz_pitch, TPBpmlz, devE, devH, devCE, psizE.x.f, psizE.y.f, 0); 

		updateSrc <<<DGsrc,DBsrc>>> ( N, devE, tstep );

		updateH <<<DGmain,DBmain,NSmain>>> ( N, TPBmain, devE, devH );
		//init_boundary_xH <<<DGinitb.x,DBinitb.x>>> ( N, devH );
		//init_boundary_yH <<<DGinitb.y,DBinitb.y>>> ( N, devH );
		init_boundary_zH <<<DGinitb.z,DBinitb.z>>> ( N, devH );
		updateCPMLxH <<<DGpml.x,DBpml.x>>> ( N, devE, devH, psixH.y.f, psixH.z.f, 0); 
		updateCPMLxH <<<DGpml.x,DBpml.x>>> ( N, devE, devH, psixH.y.b, psixH.z.b, 1); 
		updateCPMLyH <<<DGpml.y,DBpml.y>>> ( N, devE, devH, psiyH.z.f, psiyH.x.f, 0);
		updateCPMLyH <<<DGpml.y,DBpml.y>>> ( N, devE, devH, psiyH.z.b, psiyH.x.b, 1);
		updateCPMLxH <<<DGpmlx,DBpmlx>>> ( N.x, Nz_pitch, N.y*Nz_pitch, TPBpmlx, devE, devH, psixH.y.f, psixH.z.f, 0); 
		updateCPMLxH <<<DGpmlx,DBpmlx>>> ( N.x, Nz_pitch, N.y*Nz_pitch, TPBpmlx, devE, devH, psixH.y.b, psixH.z.b, 1); 
		updateCPMLyH <<<DGpmly,DBpmly>>> ( N.y, Nz_pitch, Npml*Nz_pitch, TPBpmly, devE, devH, psiyH.z.f, psiyH.x.f, 0); 
		updateCPMLyH <<<DGpmly,DBpmly>>> ( N.y, Nz_pitch, Npml*Nz_pitch, TPBpmly, devE, devH, psiyH.z.b, psiyH.x.b, 1); 
		//updateCPMLzH <<<DGpmlz,DBpmlz,NSpmlz>>> ( N, Nz_pitch, TPBpmlz, devE, devH, psizH.x.f, psizH.y.f, 0); 


		if ( tstep/10*10 == tstep ) {
			// Copy arrays from device to host
			//hipMemcpy( Ex[0][0], devE.x, (N.x+1)*N.y*N.z*sizeof(float), hipMemcpyDeviceToHost );
			hipMemcpy( Ez[0][0], devE.z, (N.x+1)*N.y*N.z*sizeof(float), hipMemcpyDeviceToHost );

			//print_array(N, Ex);
			//dumpToH5(N.x+1, N.y, N.z, N.x/2, 0, 0, N.x/2, N.y-1, N.z-1, Ex, "gpu_png/Ex-%05d.h5", tstep);
			//exec("h5topng -ZM0.1 -x0 -S4 -c /usr/share/h5utils/colormaps/dkbluered gpu_png/Ex-%05d.h5", tstep);
			dumpToH5(N.x+1, N.y, N.z, 0, 0, N.z/2, N.x, N.y-1, N.z/2, Ez, "gpu_png/Ez-%05d.h5", tstep);
			//dumpToH5(N.x+1, N.y, N.z, 0, 0, 0, N.x, N.y-1, 0, Ez, "gpu_png/Ez-%05d.h5", tstep);
			exec("h5topng -ZM0.1 -z0 -S4 -c /usr/share/h5utils/colormaps/dkbluered gpu_png/Ez-%05d.h5", tstep);

			updateTimer(t0, tstep, time_str);
			printf("tstep=%d\t%s\n", tstep, time_str);
		}
	}
	updateTimer(t0, tstep, time_str);
	printf("tstep=%d\t%s\n", tstep, time_str);
	
	free(Ex);
	free(Ez);
	freeMainArrays ( devE );
	freeMainArrays ( devH );
	freeMainArrays ( devCE );
	freePsiArrays ( psixE, psiyE, psizE );
	freePsiArrays ( psixH, psiyH, psizH );
}
