#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdarg.h>
#include <math.h>
#include <hdf5.h>


typedef struct N3 {
	int x, y, z;
} N3;


typedef struct P3F3 {
	float ***x, ***y, ***z;
} P3F3;


typedef struct P1F3 {
	float *x, *y, *z;
} P1F3;


__host__ void updateTimer(time_t t0, int tstep, char str[]) {
	int elapsedTime=(int)(time(0)-t0);
	sprintf(str, "%02d:%02d:%02d", elapsedTime/3600, elapsedTime%3600/60, elapsedTime%60);
}


__host__ void exec(char *format, ...) {
	char str[1024];
	va_list ap;
	va_start(ap, format);
	vsprintf(str, format, ap);
	system(str);
}


__host__ void dumpToH5(int Ni, int Nj, int Nk, int is, int js, int ks, int ie, int je, int ke, float ***f, char *format, ...) {
	char filename[1024];
	va_list ap;
	va_start(ap, format);
	vsprintf(filename, format, ap);
	hid_t file, dataset, filespace, memspace;

	hsize_t dimsm[3] = { Ni, Nj, Nk };
	hsize_t start[3] = { is, js, ks };
	hsize_t count[3] = { 1-is+ie, 1-js+je, 1-ks+ke };
	memspace = H5Screate_simple(3, dimsm, 0);
	filespace = H5Screate_simple(3, count, 0);
	file = H5Fcreate(filename, H5F_ACC_TRUNC, H5P_DEFAULT, H5P_DEFAULT);
	dataset = H5Dcreate(file, "Data", H5T_NATIVE_FLOAT, filespace, H5P_DEFAULT);
	H5Sselect_hyperslab(memspace, H5S_SELECT_SET, start, 0, count, 0);
	H5Dwrite(dataset, H5T_NATIVE_FLOAT, memspace, filespace, H5P_DEFAULT, f[0][0]);
	H5Dclose(dataset);
	H5Sclose(filespace);
	H5Sclose(memspace);
	H5Fclose(file);
}


__host__ void print_array(N3 N, float ***a) {
	int j,k;
	for (j=0; j<N.y; j++) {
		for (k=0; k<N.z; k++) {
			printf("%1.4f\t", a[N.x/2][j][k]);
		}
		printf("\n");
	}
	printf("\n");
}


__host__ float ***makeArray(N3 N) {
	float ***f;

	f = (float ***) calloc (N.x, sizeof(float **));
	f[0] = (float **) calloc (N.y*N.x, sizeof(float *));
	f[0][0] = (float *) calloc (N.z*N.y*N.x, sizeof(float));

	for (int i=0; i<N.x; i++) f[i] = f[0] + i*N.y;
	for (int i=0; i<N.y*N.x; i++) f[0][i] = f[0][0] + i*N.z;

	return f;
}


__host__ void set_geometry(N3 N, P3F3 CE) {
	int i,j,k;

	for (i=0; i<N.x; i++) {
		for (j=0; j<N.y; j++) {
			for (k=0; k<N.z; k++) {
				CE.x[i][j][k] = 0.5;
				CE.y[i][j][k] = 0.5;
				CE.z[i][j][k] = 0.5;
			}
		}
	}
}


__global__ void initArrays(N3 N, int Nzpit, P1F3 E, P1F3 H) {
	int idx;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	
	//printf("gridDim.x=%d\n",gridDim.x);
	//printf("blockIdx.x=%d, blockDim.x=%d, threadIdx.x=%d\n", blockIdx.x, blockDim.x, threadIdx.x);
	if ( idx < N.x*N.y*Nzpit ) {
		E.x[idx] = 0;
		E.y[idx] = 0;
		E.z[idx] = 0;
		H.x[idx] = 0;
		H.y[idx] = 0;
		H.z[idx] = 0;
	}
}


__global__ void updateE(N3 N, int Nzpit, N3 TPB, P1F3 E, P1F3 H, P1F3 CE) {
	int idx = (threadIdx.x + TPB.x*blockIdx.x) + ( (threadIdx.x + TPB.x*blockIdx.x)/Nzpit )*Nzpit*(TPB.y-1) + threadIdx.y*Nzpit;
	int Nyzpit = N.y*Nzpit;
	int i = idx/Nyzpit;
	int j = ( idx - i*Nyzpit )/Nzpit;
	int k = idx - i*Nyzpit - j*Nzpit;

	int sidx = threadIdx.x + (TPB.x+1)*threadIdx.y;

	//printf("[%.5d][%.5d][%.5d]\t[%.5d][%.5d]\t\t[%.2d,%.2d,%.2d]\n", blockIdx.x, threadIdx.y, threadIdx.x, sidx, idx, i, j, k);

	extern __shared__ float hs[];
	float* hx = (float*) hs;
	float* hy = (float*) &hx[(TPB.x+1)*(TPB.y+1)];
	float* hz = (float*) &hy[(TPB.x+1)*(TPB.y+1)];

	if ( i<N.x && j<N.y && k<N.z) {
		hx[sidx] = H.x[idx];
		hy[sidx] = H.y[idx];
		hz[sidx] = H.z[idx];
		if ( sidx%(TPB.x+1)==TPB.x-1 && k<N.z-1 ) {
			hx[sidx+1] = H.x[idx+1];
			hy[sidx+1] = H.y[idx+1];
		}
		if ( sidx/(TPB.x+1)==TPB.y-1 && j<N.y-1 ) {
			hx[sidx+TPB.x+1] = H.x[idx+Nzpit];
			hz[sidx+TPB.x+1] = H.z[idx+Nzpit];
		}
	}
	__syncthreads();

	if ( i<N.x && j<N.y && k<N.z) {
		//if ( j<N.y-1 && k<N.z-1 ) E.x[idx] += CE.x[idx]*( H.z[idx+Nzpit] - hz[sidx] - hy[sidx+1] + hy[sidx] );
		if ( j<N.y-1 && k<N.z-1 ) E.x[idx] += CE.x[idx]*( hz[sidx+TPB.x+1] - hz[sidx] - hy[sidx+1] + hy[sidx] );
		if ( i<N.x-1 && k<N.z-1 ) E.y[idx] += CE.y[idx]*( hx[sidx+1] - hx[sidx] - H.z[idx+Nyzpit] + hz[sidx] );
		//if ( i<N.x-1 && j<N.y-1 ) E.z[idx] += CE.z[idx]*( H.y[idx+Nyzpit] - hy[sidx] - H.x[idx+Nzpit] + hx[sidx] );
		if ( i<N.x-1 && j<N.y-1 ) E.z[idx] += CE.z[idx]*( H.y[idx+Nyzpit] - hy[sidx] - hx[sidx+TPB.x+1] + hx[sidx] );
		//if ( j<N.y-1 && k<N.z-1 ) E.x[idx] += CE.x[idx]*( hz[sidx+TPB.x+1] - hz[sidx] - hy[sidx+1] + hy[sidx] );
		//if ( i<N.x-1 && k<N.z-1 ) E.y[idx] += CE.y[idx]*( hx[sidx+1] - hx[sidx] - H.z[idx+Nyzpit] + hz[sidx] );
		//if ( i<N.x-1 && j<N.y-1 ) E.z[idx] += CE.z[idx]*( H.y[idx+Nyzpit] - hy[sidx] - hx[sidx+TPB.x+1] + hx[sidx] );
	}
}


__global__ void updateSrc(N3 N, int Nzpit, P1F3 E, int tstep) {
	int idx, ijk;

	idx = blockIdx.x*blockDim.x + threadIdx.x;
	ijk = idx*(N.y)*(Nzpit) + (N.y/2)*(Nzpit) + (N.z/2);

	//printf("idx=%d, ijk=%d\n", idx, ijk);
	//Ex[ijk] += __sinf(0.1*tstep);
	if ( idx < N.x ) {
		E.x[ijk] += sin(0.1*tstep);
	}
}


__global__ void updateH(N3 N, int Nzpit, int TPB, P1F3 E, P1F3 H) {
	int tk, idx;
	tk = threadIdx.x;
	idx = blockIdx.x*TPB + tk;

	int i,j,k;
	int Nyzpit = N.y*Nzpit;
	i = idx/Nyzpit;
	j = ( idx - i*Nyzpit )/Nzpit;
	k = idx - i*Nyzpit - j*Nzpit;

	extern __shared__ float es[];
	float* ex = (float*) es;
	float* ey = (float*) &ex[TPB+1];
	float* ez = (float*) &ey[TPB+1];

	if ( i<N.x && k<N.z) {
		ex[tk+1] = E.x[idx];
		ey[tk+1] = E.y[idx];
		ez[tk] = E.z[idx];
		if ( tk==0 && k>0 ) {
			ex[0] = E.x[idx-1];
			ey[0] = E.y[idx-1];
		}
	}
	__syncthreads();

	if ( i<N.x && k<N.z) {
		if ( j>0 && k>0 ) H.x[idx] -= 0.5*( ez[tk] - E.z[idx-Nzpit] - ey[tk+1] + ey[tk] );
		if ( i>0 && k>0 ) H.y[idx] -= 0.5*( ex[tk+1] - ex[tk] - ez[tk] + E.z[idx-Nyzpit] );
		if ( i>0 && j>0 ) H.z[idx] -= 0.5*( ey[tk+1] - E.y[idx-Nyzpit] - ex[tk+1] + E.x[idx-Nzpit] );
	}
}


int main() {
	int tstep;
	char time_str[32];
	time_t t0;

	// Set the parameters
	N3 N;
	N.x = 200;
	N.y = 200;
	N.z = 200;
	//N.y = 16;
	//N.z = 20;
	int TMAX = 10000;
	printf("N(%d,%d,%d), TMAX=%d\n", N.x, N.y, N.z, TMAX);

	// Allocate host memory
	P3F3 CE;
	CE.x = makeArray(N);
	CE.y = makeArray(N);
	CE.z = makeArray(N);
	float ***Ex;
	Ex = makeArray(N);

	// Geometry
	set_geometry(N, CE);


	// Allocate device memory
	P1F3 devE;
	P1F3 devH;
	P1F3 devCE;
	int z_size = N.z*sizeof(float);
	size_t pitch;
	hipMallocPitch ( (void**) &devE.x, &pitch, z_size, N.x*N.y );
	hipMallocPitch ( (void**) &devE.y, &pitch, z_size, N.x*N.y );
	hipMallocPitch ( (void**) &devE.z, &pitch, z_size, N.x*N.y );
	hipMallocPitch ( (void**) &devH.x, &pitch, z_size, N.x*N.y );
	hipMallocPitch ( (void**) &devH.y, &pitch, z_size, N.x*N.y );
	hipMallocPitch ( (void**) &devH.z, &pitch, z_size, N.x*N.y );
	hipMallocPitch ( (void**) &devCE.x, &pitch, z_size, N.x*N.y );
	hipMallocPitch ( (void**) &devCE.y, &pitch, z_size, N.x*N.y );
	hipMallocPitch ( (void**) &devCE.z, &pitch, z_size, N.x*N.y );
	
	// Copy arrays from host to device
	hipMemcpy2D ( devCE.x, pitch, CE.x[0][0], z_size, z_size, N.x*N.y, hipMemcpyHostToDevice );
	hipMemcpy2D ( devCE.y, pitch, CE.y[0][0], z_size, z_size, N.x*N.y, hipMemcpyHostToDevice );
	hipMemcpy2D ( devCE.z, pitch, CE.z[0][0], z_size, z_size, N.x*N.y, hipMemcpyHostToDevice );
	
	int Nz_pitch = pitch/4;
	printf("pitch= %u, Nz_pitch= %d\n", pitch, Nz_pitch);

	// Set the GPU parameters
	int Ntot = N.x*N.y*Nz_pitch;
	int TPB = 256;	// Number of threads per block
	int BPG = Ntot%TPB == 0 ? Ntot/TPB : Ntot/TPB + 1; // Number of thread blocks per grid
	dim3 Dg = dim3(BPG);
	dim3 Db = dim3(TPB);
	size_t Ns = sizeof(float)*( (TPB+1)+(TPB+1)+(TPB) );
	printf("Threads per block: %d\n", TPB);
	if ( TPB > 512 ) {
		printf("Error: An excessive number of threads per block.\n");
		exit(0);
	}
	printf("Blocks per grid: %d\n", BPG);
	if ( BPG > 65535 ) {
		printf("Error: An excessive number of blocks per grid.\n");
		exit(0);
	}
	printf("Number of bytes in shared memory: %d\n", Ns);

	N3 TPBmain; 
	TPBmain.x = 128;
	TPBmain.y = 4;
	TPBmain.z = 1;
	int BPGmain = Ntot%(TPBmain.x*TPBmain.y) == 0 ? Ntot/(TPBmain.x*TPBmain.y) : Ntot/(TPBmain.x*TPBmain.y) + 1; // Number of thread blocks per grid
	//BPGmain.x = Nz_pitch%TPBmain.x == 0 ? Nz_pitch/TPBmain.x : Nz_pitch/TPBmain.x + 1; // Number of thread blocks per grid
	//BPGmain.y = (N.x*N.y)%TPBmain.y == 0 ? (N.x*N.y)/TPBmain.y : (N.x*N.y)/TPBmain.y + 1; 
	//dim3 Dgmain = dim3(BPGmain.x, BPGmain.y);
	dim3 Dgmain = dim3(BPGmain);
	dim3 Dbmain = dim3(TPBmain.x, TPBmain.y, TPBmain.z);
	//size_t Nsmain = sizeof(float)*( (TPBmain.x+1)*(TPBmain.y+1) + (TPBmain.x+1)*TPBmain.y + TPBmain.x*(TPBmain.y+1) );
	size_t Nsmain = sizeof(float)*( 3*(TPBmain.x+1)*(TPBmain.y+1) );

	int TPBsrc = N.x;
	int BPGsrc = 1;
	dim3 Dgsrc(BPGsrc);
	dim3 Dbsrc(TPBsrc);

	int TPBinit = Nz_pitch;
	int BPGinit = Ntot%TPBinit == 0 ? Ntot/TPBinit : Ntot/TPBinit + 1;
	dim3 Dginit(BPGinit);
	dim3 Dbinit(TPBinit);

	/*
	// Initialize the device arrays
	initArrays <<<Dginit,Dbinit>>> ( N, Nz_pitch, devE, devH );

	// Main time loop
	t0 = time(0);
	//for ( tstep=1; tstep<=TMAX; tstep++) {
	for ( tstep=1; tstep<=200; tstep++) {
		// Update on the GPU
		//updateE <<<Dg,Db,Ns>>> ( N, Nz_pitch, TPB, devE, devH, devCE );
		updateE <<<Dgmain,Dbmain,Nsmain>>> ( N, Nz_pitch, TPBmain, devE, devH, devCE );
		updateSrc <<<Dgsrc,Dbsrc>>> ( N, Nz_pitch, devE, tstep );
		updateH <<<Dg,Db,Ns>>> ( N, Nz_pitch, TPB, devE, devH );

		if ( tstep/10*10 == tstep ) {
			// Copy arrays from device to host
			hipMemcpy2D( Ex[0][0], z_size, devE.x, pitch, z_size, N.x*N.y, hipMemcpyDeviceToHost );

			//print_array(N, Ex);
			dumpToH5(N.x, N.y, N.z, N.x/2, 0, 0, N.x/2, N.y-1, N.z-1, Ex, "gpu_png/Ex-%05d.h5", tstep);
			exec("h5topng -ZM0.1 -x0 -S4 -c /usr/share/h5utils/colormaps/dkbluered gpu_png/Ex-%05d.h5", tstep);

			updateTimer(t0, tstep, time_str);
			printf("tstep=%d\t%s\n", tstep, time_str);
		}
	}
	updateTimer(t0, tstep, time_str);
	printf("tstep=%d\t%s\n", tstep, time_str);
	*/
	for ( tstep=1; tstep<=10; tstep++ ) updateE <<<Dgmain,Dbmain,Nsmain>>> ( N, Nz_pitch, TPBmain, devE, devH, devCE );
	//for ( tstep=1; tstep<=10; tstep++ ) updateH <<<Dg,Db,Ns>>> ( N, Nz_pitch, TPB, devE, devH );
}
