#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <stdarg.h>
#include <math.h>
#include <hdf5.h>

#define TPBx 16	// Number of threads per block
#define TPBy 4
#define TPBz 4


typedef struct N3 {
	int x, y, z;
} N3;


typedef struct P3F3 {
	float ***x, ***y, ***z;
} P3F3;


typedef struct P1F3 {
	float *x, *y, *z;
} P1F3;


__host__ void updateTimer(time_t t0, int tstep, char str[]) {
	int elapsedTime=(int)(time(0)-t0);
	sprintf(str, "%02d:%02d:%02d", elapsedTime/3600, elapsedTime%3600/60, elapsedTime%60);
}


__host__ void exec(char *format, ...) {
	char str[1024];
	va_list ap;
	va_start(ap, format);
	vsprintf(str, format, ap);
	system(str);
}


__host__ void dumpToH5(int Ni, int Nj, int Nk, int is, int js, int ks, int ie, int je, int ke, float ***f, char *format, ...) {
	char filename[1024];
	va_list ap;
	va_start(ap, format);
	vsprintf(filename, format, ap);
	hid_t file, dataset, filespace, memspace;

	hsize_t dimsm[3] = { Ni, Nj, Nk };
	hsize_t start[3] = { is, js, ks };
	hsize_t count[3] = { 1-is+ie, 1-js+je, 1-ks+ke };
	memspace = H5Screate_simple(3, dimsm, 0);
	filespace = H5Screate_simple(3, count, 0);
	file = H5Fcreate(filename, H5F_ACC_TRUNC, H5P_DEFAULT, H5P_DEFAULT);
	dataset = H5Dcreate(file, "Data", H5T_NATIVE_FLOAT, filespace, H5P_DEFAULT);
	H5Sselect_hyperslab(memspace, H5S_SELECT_SET, start, 0, count, 0);
	H5Dwrite(dataset, H5T_NATIVE_FLOAT, memspace, filespace, H5P_DEFAULT, f[0][0]);
	H5Dclose(dataset);
	H5Sclose(filespace);
	H5Sclose(memspace);
	H5Fclose(file);
}


__host__ void print_array(N3 N, float ***a) {
	int j,k;
	for (j=0; j<N.y; j++) {
		for (k=0; k<N.z; k++) {
			printf("%1.4f\t", a[N.x/2][j][k]);
		}
		printf("\n");
	}
	printf("\n");
}


__host__ float ***makeArray(N3 N) {
	float ***f;

	f = (float ***) calloc (N.x, sizeof(float **));
	f[0] = (float **) calloc (N.y*N.x, sizeof(float *));
	f[0][0] = (float *) calloc (N.z*N.y*N.x, sizeof(float));

	for (int i=0; i<N.x; i++) f[i] = f[0] + i*N.y;
	for (int i=0; i<N.y*N.x; i++) f[0][i] = f[0][0] + i*N.z;

	return f;
}


__host__ void set_geometry(N3 N, P3F3 CE) {
	int i,j,k;

	for (i=0; i<N.x; i++) {
		for (j=0; j<N.y; j++) {
			for (k=0; k<N.z; k++) {
				CE.x[i][j][k] = 0.5;
				CE.y[i][j][k] = 0.5;
				CE.z[i][j][k] = 0.5;
			}
		}
	}
}


__global__ void initArrays(N3 N, int Nzpit, P1F3 E, P1F3 H) {
	int idx;
	idx = blockIdx.x*blockDim.x + threadIdx.x;
	
	//printf("gridDim.x=%d\n",gridDim.x);
	//printf("blockIdx.x=%d, blockDim.x=%d, threadIdx.x=%d\n", blockIdx.x, blockDim.x, threadIdx.x);
	if ( idx < N.x*N.y*Nzpit ) {
		E.x[idx] = 0;
		E.y[idx] = 0;
		E.z[idx] = 0;
		H.x[idx] = 0;
		H.y[idx] = 0;
		H.z[idx] = 0;
	}
}


__global__ void updateE(N3 N, int Nzpit, N3 BPG, P1F3 E, P1F3 H, P1F3 CE) {
	int bk;
	int tk, tj, ti;
	int i, j, k;
	bk = blockIdx.x;
	tk = threadIdx.x;
	tj = threadIdx.y;
	ti = threadIdx.z;
	k = TPBx*( bk%BPG.x ) + tk;
	j = TPBy*( (bk/BPG.x)%BPG.y ) + tj;
	i = TPBz*( bk/(BPG.x*BPG.y) ) + ti;

	//printf("bk(%d),\tbk%BPGx (%d),\t(bk/BPGx)%BPGy (%d),\tbk/(BPGx*BPGy) (%d)\n", bk, bk%BPG.x, (bk/BPG.x)%BPG.y, bk/(BPG.x*BPG.y) );
	//printf("blockIdx(%d),\tthreadIdx(%d,%d,%d),\tkji(%d,%d,%d)\n", bk, tk, tj, ti, k, j, i);

	int Nyzpit = N.y*Nzpit;
	int idx = k + Nzpit*j + Nyzpit*i;
	
	__shared__ float hx[TPBz][TPBy+1][TPBx+1];
	__shared__ float hy[TPBz+1][TPBy][TPBx+1];
	__shared__ float hz[TPBz+1][TPBy+1][TPBx];

	if ( i<N.x && j<N.y && k<N.z ) {
		//printf("(%d),\t(%d,%d,%d),\t(%d,%d,%d),\t%d\n", bk, tk, tj, ti, k, j, i, idx);

		hx[ti][tj][tk] = H.x[idx];
		hy[ti][tj][tk] = H.y[idx];
		hz[ti][tj][tk] = H.z[idx];

		if ( tk==TPBx-1 && k<N.z-1 ) {
			hx[ti][tj][tk+1] = H.x[idx+1];
			hy[ti][tj][tk+1] = H.y[idx+1];
		}
		if ( tj==TPBy-1 && j<N.y-1 ) {
			hx[ti][tj+1][tk] = H.x[idx+Nzpit];
			hz[ti][tj+1][tk] = H.z[idx+Nzpit];
		}
		if ( ti==TPBz-1 && i<N.x-1 ) {
			hy[ti+1][tj][tk] = H.y[idx+Nyzpit];
			hz[ti+1][tj][tk] = H.z[idx+Nyzpit];
		}
	}
	__syncthreads();

	if ( i<N.x && j<N.y && k<N.z ) {
		if ( j<N.y-1 && k<N.z-1 ) {
			//if ( j==8 && k==10 ) printf("Ex[%d,8,10]=%g\n", i, E.x[idx]);
				E.x[idx] += CE.x[idx]*( hz[ti][tj+1][tk] - hz[ti][tj][tk] - hy[ti][tj][tk+1] + hy[ti][tj][tk] );
			}
		if ( i<N.x-1 && k<N.z-1 ) E.y[idx] += CE.y[idx]*( hx[ti][tj][tk+1] - hx[ti][tj][tk] - hz[ti+1][tj][tk] + hz[ti][tj][tk] );
		if ( i<N.x-1 && j<N.y-1 ) E.z[idx] += CE.z[idx]*( hy[ti+1][tj][tk] - hy[ti][tj][tk] - hx[ti][tj+1][tk] + hx[ti][tj][tk] );
	}
}


__global__ void updateSrc(N3 N, int Nzpit, P1F3 E, int tstep) {
	int idx, ijk;

	idx = blockIdx.x*blockDim.x + threadIdx.x;
	ijk = idx*(N.y)*(Nzpit) + (N.y/2)*(Nzpit) + (N.z/2);

	//printf("idx=%d, ijk=%d\n", idx, ijk);
	//Ex[ijk] += __sinf(0.1*tstep);
	if ( idx < N.x ) {
		E.x[ijk] += sin(0.1*tstep);
	}
}


__global__ void updateH(N3 N, int Nzpit, N3 BPG, P1F3 E, P1F3 H) {
	int bk;
	int tk, tj, ti;
	int i, j, k;
	bk = blockIdx.x;
	tk = threadIdx.x;
	tj = threadIdx.y;
	ti = threadIdx.z;
	k = TPBx*( bk%BPG.x ) + tk;
	j = TPBy*( (bk/BPG.x)%BPG.y ) + tj;
	i = TPBz*( bk/(BPG.x*BPG.y) ) + ti;

	int Nyzpit = N.y*Nzpit;
	int idx = k + Nzpit*j + Nyzpit*i;
	
	__shared__ float ex[TPBz][TPBy+1][TPBx+1];
	__shared__ float ey[TPBz+1][TPBy][TPBx+1];
	__shared__ float ez[TPBz+1][TPBy+1][TPBx];

	if ( i<N.x && j<N.y && k<N.z ) {
		ex[ti][tj+1][tk+1] = E.x[idx];
		ey[ti+1][tj][tk+1] = E.y[idx];
		ez[ti+1][tj+1][tk] = E.z[idx];
		if ( tk==0 && k>0 ) {
			ex[ti][tj+1][0] = E.x[idx-1];
			ey[ti+1][tj][0] = E.y[idx-1];
		}
		if ( tj==0 && j>0 ) {
			ex[ti][0][tk+1] = E.x[idx-Nzpit];
			ez[ti+1][0][tk] = E.z[idx-Nzpit];
		}
		if ( ti==0 && i>0 ) {
			ey[0][tj][tk+1] = E.y[idx-Nyzpit];
			ez[0][tj+1][tk] = E.z[idx-Nyzpit];
		}
	}
	__syncthreads();

	if ( i<N.x && j<N.y && k<N.z ) {
		if ( j>0 && k>0 ) H.x[idx] -= 0.5*( ez[ti+1][tj+1][tk] - ez[ti+1][tj][tk] - ey[ti+1][tj][tk+1] + ey[ti+1][tj][tk] );
		if ( i>0 && k>0 ) H.y[idx] -= 0.5*( ex[ti][tj+1][tk+1] - ex[ti][tj+1][tk] - ez[ti+1][tj+1][tk] + ez[ti][tj+1][tk] );
		/*
			if ( j==8 && k==10 ) {
				printf("[%.2d]\t\t[%.2d,%.2d,%.2d]\t\t[%.2d,%.2d,%.2d]\t\t[%d]\n", bk, ti, tj, tk, i, j, k, idx);
				//printf("Ex[%d,%d,%d]=%g\n", i, j, k, E.x[idx]);
				printf("ex[%d,%d,%d]=%g\n", ti, tj+1, tk+1, ex[ti][tj+1][tk+1]);
				printf("ex[%d,%d,%d]=%g\n", ti, tj+1, tk, ex[ti][tj+1][tk]);
				printf("Hy[%d][%d,%d,%d]=%g\n", idx, i, j, k, H.y[idx] );
				printf("\n");
			}
		}
		*/
		if ( i>0 && j>0 ) H.z[idx] -= 0.5*( ey[ti+1][tj][tk+1] - ey[ti][tj][tk+1] - ex[ti][tj+1][tk+1] + ex[ti][tj][tk+1] );
	}
}


int main() {
	int tstep;
	char time_str[32];
	time_t t0;

	// Set the parameters
	N3 N;
	N.x = 100;
	N.y = 200;
	N.z = 500;
	//N.y = 16;
	//N.z = 20;
	int TMAX = 1000;
	printf("N(%d,%d,%d), TMAX=%d\n", N.x, N.y, N.z, TMAX);

	// Allocate host memory
	P3F3 CE;
	CE.x = makeArray(N);
	CE.y = makeArray(N);
	CE.z = makeArray(N);

	/*
	float ***Ex;
	P3F3 H;
	Ex = makeArray(N);
	H.y = makeArray(N);
	H.z = makeArray(N);
	*/

	// Geometry
	set_geometry(N, CE);


	// Allocate device memory
	P1F3 devE;
	P1F3 devH;
	P1F3 devCE;
	int z_size = N.z*sizeof(float);
	size_t pitch;
	hipMallocPitch ( (void**) &devE.x, &pitch, z_size, N.x*N.y );
	hipMallocPitch ( (void**) &devE.y, &pitch, z_size, N.x*N.y );
	hipMallocPitch ( (void**) &devE.z, &pitch, z_size, N.x*N.y );
	hipMallocPitch ( (void**) &devH.x, &pitch, z_size, N.x*N.y );
	hipMallocPitch ( (void**) &devH.y, &pitch, z_size, N.x*N.y );
	hipMallocPitch ( (void**) &devH.z, &pitch, z_size, N.x*N.y );
	hipMallocPitch ( (void**) &devCE.x, &pitch, z_size, N.x*N.y );
	hipMallocPitch ( (void**) &devCE.y, &pitch, z_size, N.x*N.y );
	hipMallocPitch ( (void**) &devCE.z, &pitch, z_size, N.x*N.y );
	
	// Copy arrays from host to device
	hipMemcpy2D ( devCE.x, pitch, CE.x[0][0], z_size, z_size, N.x*N.y, hipMemcpyHostToDevice );
	hipMemcpy2D ( devCE.y, pitch, CE.y[0][0], z_size, z_size, N.x*N.y, hipMemcpyHostToDevice );
	hipMemcpy2D ( devCE.z, pitch, CE.z[0][0], z_size, z_size, N.x*N.y, hipMemcpyHostToDevice );
	
	int Nz_pitch = pitch/4;
	printf("pitch= %u, Nz_pitch= %d\n", pitch, Nz_pitch);

	// Set the GPU parameters
	N3 BPG;	// Number of thread blocks per grid
	BPG.x = Nz_pitch/TPBx;
	BPG.y = N.y%TPBy == 0 ? N.y/TPBy : N.y/TPBy + 1;
	BPG.z = N.x%TPBz == 0 ? N.x/TPBz : N.x/TPBz + 1;
	dim3 Dg = dim3(BPG.x*BPG.y*BPG.z);
	dim3 Db = dim3(TPBx, TPBy, TPBz);
	//dim3 Dg = dim3(20);
	//dim3 Db = dim3(16,3,4);
	printf("Threads per block: %d (%d,%d,%d)\n", TPBx*TPBy*TPBz, TPBx, TPBy, TPBz);
	if ( TPBx*TPBy*TPBz > 512 ) {
		printf("Error: An excessive number of threads per block.\n");
		exit(0);
	}
	printf("Blocks per grid: %d (%d,%d,%d)\n", BPG.x*BPG.y*BPG.z, BPG.x, BPG.y, BPG.z);
	if ( BPG.x*BPG.y*BPG.z > 65535 ) {
		printf("Error: An excessive number of blocks per grid.\n");
		exit(0);
	}

	int TPBsrc = N.x;
	int BPGsrc = 1;
	dim3 Dgsrc(BPGsrc);
	dim3 Dbsrc(TPBsrc);

	int Ntot = N.x*N.y*Nz_pitch;
	int TPBinit = Nz_pitch;
	int BPGinit = Ntot%TPBinit == 0 ? Ntot/TPBinit : Ntot/TPBinit + 1;
	dim3 Dginit(BPGinit);
	dim3 Dbinit(TPBinit);

	// Initialize the device arrays
	initArrays <<<Dginit,Dbinit>>> ( N, Nz_pitch, devE, devH );

	// Main time loop
	t0 = time(0);
	//for ( tstep=1; tstep<=TMAX; tstep++) {
	for ( tstep=1; tstep<=10; tstep++) {
		// Update on the GPU
		updateE <<<Dg,Db>>> ( N, Nz_pitch, BPG, devE, devH, devCE );
		//updateSrc <<<Dgsrc,Dbsrc>>> ( N, Nz_pitch, devE, tstep );
		//updateH <<<Dg,Db>>> ( N, Nz_pitch, BPG, devE, devH );

		/*
		//if ( tstep/10*10 == tstep ) {
			// Copy arrays from device to host
			hipMemcpy2D( Ex[0][0], z_size, devE.x, pitch, z_size, N.x*N.y, hipMemcpyDeviceToHost );
			//hipMemcpy2D( H.y[0][0], z_size, devH.y, pitch, z_size, N.x*N.y, hipMemcpyDeviceToHost );
			//hipMemcpy2D( H.z[0][0], z_size, devH.z, pitch, z_size, N.x*N.y, hipMemcpyDeviceToHost );

			//printf("Ex\n");
			//print_array(N, Ex);
			//printf("Hy\n");
			//print_array(N, H.y);
			//printf("Hz\n");
			//print_array(N, H.z);
			dumpToH5(N.x, N.y, N.z, N.x/2, 0, 0, N.x/2, N.y-1, N.z-1, Ex, "gpu_png/Ex-%05d.h5", tstep);
			exec("h5topng -ZM0.1 -x0 -S4 -c /usr/share/h5utils/colormaps/dkbluered gpu_png/Ex-%05d.h5", tstep);

			updateTimer(t0, tstep, time_str);
			printf("tstep=%d\t%s\n", tstep, time_str);
		//}
		*/
	}
	updateTimer(t0, tstep, time_str);
	printf("tstep=%d\t%s\n", tstep, time_str);
}
