
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>


__global__ void kern(float *a) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	a[idx] = a[idx] + 3.0;
}


int main() {
	int tn, nx=32*16*2;
	
	float *a, *b, *c;
	a = (float *) malloc (nx*sizeof(float));
	b = (float *) malloc (nx*sizeof(float));
	c = (float *) malloc (nx*sizeof(float));

	float *a_gpu, *b_gpu, *c_gpu;
	int size = nx*sizeof(float);
	hipMalloc ( (void**) &a_gpu, size );
	hipMalloc ( (void**) &b_gpu, size );
	hipMalloc ( (void**) &c_gpu, size );

	hipMemcpy ( a_gpu, a, size, hipMemcpyHostToDevice );
	hipMemcpy ( b_gpu, b, size, hipMemcpyHostToDevice );
	hipMemcpy ( c_gpu, c, size, hipMemcpyHostToDevice );
	
	int tpb = 512;
	int bpg = nx/tpb;

	for(tn=1; tn<=10; tn++) {
		kern <<<dim3(bpg),dim3(tpb)>>> (a_gpu);
	}
}
