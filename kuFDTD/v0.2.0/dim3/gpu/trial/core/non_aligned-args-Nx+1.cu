
#include <hip/hip_runtime.h>
__global__ void update_e( int Nz, int Nyz, int Nyzm, float *Ex, float *Ey, float *Ez, float *Hx, float *Hy, float *Hz, float *CEx, float *CEy, float *CEz ) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int fidx = idx + idx/(Nz-1) + idx/Nyzm*Nz + Nyz + Nz + 1; 
	
	Ex[fidx] += CEx[fidx]*( Hz[fidx+Nz] - Hz[fidx] - Hy[fidx+1] + Hy[fidx] );
	Ey[fidx] += CEy[fidx]*( Hx[fidx+1] - Hx[fidx] - Hz[fidx+Nyz] + Hz[fidx] );
	Ez[fidx] += CEz[fidx]*( Hy[fidx+Nyz] - Hy[fidx] - Hx[fidx+Nz] + Hx[fidx] );
}


__global__ void update_h( int Nz, int Nyz, int Nyzm, float *Ex, float *Ey, float *Ez, float *Hx, float *Hy, float *Hz ) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int fidx = idx + idx/(Nz-1) + idx/Nyzm*Nz + Nyz + Nz + 1; 

	Hx[fidx] -= 0.5*( Ez[fidx] - Ez[fidx-Nz] - Ey[fidx] + Ey[fidx-1] );
	Hy[fidx] -= 0.5*( Ex[fidx] - Ex[fidx-1] - Ez[fidx] + Ez[fidx-Nyz] );
	Hz[fidx] -= 0.5*( Ey[fidx] - Ey[fidx-Nyz] - Ex[fidx] + Ex[fidx-Nz] );
}
