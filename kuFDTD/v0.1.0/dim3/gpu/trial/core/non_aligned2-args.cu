
#include <hip/hip_runtime.h>
__global__ void update_e( int Nz, int Nyz, int Nzm, int Nyzm, int Nxyzm, float *Ex, float *Ey, float *Ez, float *Hx, float *Hy, float *Hz, float *CEx, float *CEy, float *CEz ) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int fidx = idx + idx/Nzm*2 + idx/Nyzm*Nz*2 + Nyz + Nz + 1; 
	
	if ( fidx < Nxyzm ) {
		Ex[fidx] += CEx[fidx]*( Hz[fidx+Nz] - Hz[fidx] - Hy[fidx+1] + Hy[fidx] );
		Ey[fidx] += CEy[fidx]*( Hx[fidx+1] - Hx[fidx] - Hz[fidx+Nyz] + Hz[fidx] );
		Ez[fidx] += CEz[fidx]*( Hy[fidx+Nyz] - Hy[fidx] - Hx[fidx+Nz] + Hx[fidx] );
	}
}


__global__ void update_h( int Nz, int Nyz, int Nzm, int Nyzm, int Nxyzm, float *Ex, float *Ey, float *Ez, float *Hx, float *Hy, float *Hz ) {
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	int fidx = idx + idx/Nzm*2 + idx/Nyzm*Nz*2 + Nyz + Nz + 1; 

	if ( fidx < Nxyzm ) {
		Hx[fidx] -= 0.5*( Ez[fidx] - Ez[fidx-Nz] - Ey[fidx] + Ey[fidx-1] );
		Hy[fidx] -= 0.5*( Ex[fidx] - Ex[fidx-1] - Ez[fidx] + Ez[fidx-Nyz] );
		Hz[fidx] -= 0.5*( Ey[fidx] - Ey[fidx-Nyz] - Ex[fidx] + Ex[fidx-Nz] );
	}
}