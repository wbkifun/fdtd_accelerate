
#include <hip/hip_runtime.h>
__global__ void update_e( int Ny, int Nz, float *Ex, float *Ey, float *Ez, float *Hx, float *Hy, float *Hz, float *CEx, float *CEy, float *CEz ) {
	int tk = threadIdx.x;
	int idx = blockIdx.x*blockDim.x + tk;
	int Nyz = Ny*Nz;
	//int fidx = idx + idx/(Nz-1) + Nyz + Nz + 1; 
	int fidx = idx + idx/(Nz-1) + idx/( (Nz-1)*(Ny-1) )*(Nz-1) + Nyz + Nz + 1; 
	
	extern __shared__ float hs[];
	float* hx = (float*) hs;
	float* hy = (float*) &hx[blockDim.x+1];
	float* hz = (float*) &hy[blockDim.x+1];

	hx[tk] = Hx[fidx];
	hy[tk] = Hy[fidx];
	hz[tk] = Hz[fidx];
	
	if ( tk==blockDim.x-1 ) {
		hx[tk+1] = Hx[fidx+1];
		hy[tk+1] = Hy[fidx+1];
	}
	__syncthreads();
	
	Ex[fidx] += CEx[fidx]*( Hz[fidx+Nz] - hz[tk] - hy[tk+1] + hy[tk] );
	Ey[fidx] += CEy[fidx]*( hx[tk+1] - hx[tk] - Hz[fidx+Nyz] + hz[tk] );
	Ez[fidx] += CEz[fidx]*( Hy[fidx+Nyz] - hy[tk] - Hx[fidx+Nz] + hx[tk] );
}


__global__ void update_h( int Ny, int Nz, float *Ex, float *Ey, float *Ez, float *Hx, float *Hy, float *Hz ) {
	int tk = threadIdx.x;
	int idx = blockIdx.x*blockDim.x + tk;
	int Nyz = Ny*Nz;
	//int fidx = idx + idx/(Nz-1) + Nyz + Nz + 1; 
	int fidx = idx + idx/(Nz-1) + idx/( (Nz-1)*(Ny-1) )*(Nz-1) + Nyz + Nz + 1; 
	
	extern __shared__ float es[];
	float* ex = (float*) es;
	float* ey = (float*) &ex[blockDim.x+1];
	float* ez = (float*) &ey[blockDim.x+1];

	ex[tk+1] = Ex[fidx];
	ey[tk+1] = Ey[fidx];
	ez[tk]   = Ez[fidx];
	if ( tk==0 ) {
		ex[0] = Ex[fidx-1];
		ey[0] = Ey[fidx-1];
	}
	__syncthreads();
	
	Hx[fidx] -= 0.5*( ez[tk] - Ez[fidx-Nz] - ey[tk+1] + ey[tk] );
	Hy[fidx] -= 0.5*( ex[tk+1] - ex[tk] - ez[tk] + Ez[fidx-Nyz] );
	Hz[fidx] -= 0.5*( ey[tk+1] - Ey[fidx-Nyz] - ex[tk+1] + Ex[fidx-Nz] );
}